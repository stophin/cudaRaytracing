#include "hip/hip_runtime.h"

#define RUN_DEVICE

//#define WIN_DEBUG
#ifdef WIN_DEBUG
#define DEBUG(Arg, ...) printf(Arg, ##__VA_ARGS__)
#else
#define DEBUG(x, ...)
#endif

#define MAX_PRECISE 100000.0

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "math3d/Manager3D.h"
#include "math3d/Texture3D.h"
#include "math3d/Texture.h"
#include "platform/Device.h"
#include "raytracing/Ray.h"
#include "common/MultiLink.h"

_PLATFORM Manager3D * dev_man;
_PLATFORM TextureManager * dev_tman;
_PLATFORM VertsMan** dev_vman;
_PLATFORM COLORREF * dev_res;
_PLATFORM EFTYPE * dev_resf;
_PLATFORM Device * _device;
_PLATFORM DWORD * tango;
_PLATFORM EFTYPE * depth;
__device__ int *a;
#define THREAD_W		50
#define THREAD_H		50
#define THREAD_W_R		30
#define THREAD_H_R		30
#define MAX_ITERATOR	THREAD_W_R * THREAD_H_R
#define WIN_WIDTH	800
#define WIN_HEIGHT	600
_PLATFORM Obj3D ** objIterator;
_PLATFORM VertsPoolImp * verts_pool;
_PLATFORM OctPoolImp * octPoolImp;
#ifndef RUN_DEVICE
Manager3D man;
VObjPoolImp vobjPoolImp;
ObjPoolImp objPoolImp;
CamPoolImp camPoolImp;
LgtPoolImp lgtPoolImp;
OctPoolImp _octPoolImp;
Group3DPoolImp group3DPoolImp;
#endif
typedef struct Triangles {
	VObj * v;
	VObj * v0;
	VObj * v1;
};
_PLATFORM Triangles * tgIterator;
Device device;

__global__ void renderRayTracing(EFTYPE * res, INT size, Manager3D * _man, VertsMan** _vman, Triangles * tgIterator, INT grid, INT iteratorW, INT iteratorH, Device * device, VertsPoolImp * vertsPool)
{

#ifdef RUN_DEVICE
	int res_index = 0;

	Manager3D& man = *_man;
	Obj3D * obj = NULL;
	Cam3D * cam = NULL;

	INT iteratorIndex = 0;
	int sx = blockIdx.x * iteratorW;
	int sy = blockIdx.y * iteratorH;
	int line = 0; //block lines
	int ex = blockIdx.x * iteratorW + iteratorW - line;
	int ey = blockIdx.y * iteratorH + iteratorH - line;
	//int tid = blockIdx.x  * blockDim.x + blockIdx.y;
	int tid = blockIdx.x + blockIdx.y * gridDim.x;
	//int tid = threadIdx.x + blockIdx.x * blockDim.x;

	cam = man.cams.link;
	res[99] = (DWORD)cam;
	res[98] = 1000 + tid;
	res[97] = (DWORD)man.octTree.link;
	res[96] = man.octTree.linkcount;
	res[0] = sx;
	res[1] = sy;
	res[2] = ex;
	res[3] = ey;

	Verts vertsTemp;

	Lgt3D * lgt;
	EFTYPE f;
	Vert3D n0, n1, n2, n3, p;
	Vert3D _n0, _n1, _n2, _n3;
	EPoint l1, l0, l;
	EFTYPE z;
	Ray ray;
	INT index;
	EFTYPE _i, _j;
	INT _index;
	DWORD * _raytracing;
	EFTYPE trans;
	if (tid >= device->threadImageCount) {
		//return;
	}
	//DWORD *_image = device->threadImage[tid];
	//memset(_image, 0, sizeof(DWORD)* device->width * device->height);

	VertsPoolImp * pool = &vertsPool[tid];
	memset(pool, 0, sizeof(VertsPoolImp));
	_VertsPoolImp(pool);
	VertsMan& raytracing_verts = *_vman[tid * 2 + 0];
	VertsMan& raytracing_verts_accumulated = *_vman[tid * 2 + 1];
	//VertsMan raytracing_verts;
	//VertsMan* praytracing_verts = NULL;
	//hipMalloc(&praytracing_verts, sizeof(VertsMan));
	//VertsMan& raytracing_verts = *praytracing_verts;
	_VertsMan(&raytracing_verts, 0, pool);
	////VertsMan raytracing_verts_accumulated;
	//VertsMan* praytracing_verts_accumulated = NULL;
	//hipMalloc(&praytracing_verts_accumulated, sizeof(VertsMan));
	//VertsMan& raytracing_verts_accumulated = *praytracing_verts_accumulated;
	_VertsMan(&raytracing_verts_accumulated, 1, pool);

	VObjMan * link = NULL;
	ObjMan * olink;
	//MultiLinkList<Obj3D> octs(MAX_OBJ3D_LINK + 1 + id);
	//ObjMan octs;
	DWORD * __image;
	//reflection times
	INT count, shadow_count;
	//for each pixel in width * height's screen
	for (int y = sy; y < ey; y++) {
		for (int x = sx; x < ex; x++) {
			index = y * device->width + x;
			if (index > device->width * device->height) {
				continue;
			}
			//_raytracing = &device->raytracing[index];
			//_raytracing = &_image[index];
			//_raytracing = &device->raytracing[index];
			_raytracing = &device->tango[index];
			//*_raytracing = WHITE;
			//Orthographic
			if (cam->type == 1) {
				//get original vert from this pixel
				n0.set((x - cam->offset_w) / cam->scale_w, (y - cam->offset_h) / cam->scale_h, 0, 1);
				//get direction vert
				n1.set(cam->lookat).negative();
				n1.normalize().negative();
				//set ray
				ray.set(n0, n1);
				//set ray type
				ray.type = 0;
			}
			//Oblique
			else if (cam->type == 2) {
				//get original vert from this pixel
				n0.set(0, 0, 0, 1);
				//get direction vert
				n2.set((x - cam->offset_w) / cam->scale_w, (y - cam->offset_h) / cam->scale_h, 0, 1);
				cam->anti_normalize(cam, n2, cam->znear);
				//n0.set(n2);
				n1.set(cam->lookat) * cam->znear;
				n1 + n2;
				n1.w = 1;
				n1.normalize().negative();
				//set ray
				ray.set(n0, n1);
				//set ray type
				ray.type = 0;
			}

			Verts * nearest_vert_shadow = NULL;
			Lgt3D * cur_lgt = man.lgts.link;
			shadow_count = 0;
			//ray tracing depth
			count = 3;
			do {
				// when the ray is reflection or refraction
				// use the objects around instead of all the objects

				if (0 && (1 == ray.type || 2 == ray.type)) {
					if (!ray.obj) {
						ray.obj = ray.obj;
					}
					//olink = &octs;
					//olink = &man.octs;
					olink->clearLink(olink);
					man.octTree.CollisionA(&man.octTree, man.octTree.link, (Obj3D*)ray.obj, olink);
				}
				else {
					olink = &man.objs;
				}

				Obj3D * obj = olink->link;
				if (obj) {
					int render_state = 0;
					VObj * v, *v0, *v1, *vtemp;

					EFTYPE trans_last = 1000;
					// for each triangle
					do {
						//object aabb intersection
						INT intersection = 1;
						if (&man.objs == olink) {
							//intersection = man.octTree.Collision(&man.octTree, man.octTree.link, ray.original, ray.direction, (Camera3D*)cam, obj);
							intersection = Collision(ray.original, ray.direction, obj, trans_last);
							//intersection = 1;
						}
						if (intersection) {

							// when the ray is reflection
							// then use all the verts instead 
							// of the verts after frustrum culling
							if (1 == ray.type) {
								link = &obj->verts;
							}
							else {
								link = &obj->verts_r;
							}
							v = link->link;
							// more than 3 verts
							if (v && link->linkcount >= 3) {
								v0 = NULL; v1 = NULL;
								int traverseCount = 0;
								do {
									traverseCount++;
									//there must be three verts
									if (v0 && v1) {
										// back face culling
										// when the ray is reflection or shadow testing
										// then do not need back face culling
										if ((v->backface > 0 || v->backface <= 0) || 1 == ray.type || (3 == ray.type && obj->backfaceculling == 0))
										{
											//NOTE: ray tracing is in camera coordinate
											//get intersect point
											trans = Vert3D::GetLineIntersectPointWithTriangle(v->v_c, v0->v_c, v1->v_c, ray.original, ray.direction, trans_last, p);
											//trans = 1;
											//trans is greater than zero, and less than last trans
											if (EP_GTZERO(trans)) {
												//RAYTRACING_MUTEX(Verts * verts = new Verts(););
												//Verts vertsTemp;
												Verts* verts = pool->vertsPool.get(&pool->vertsPool);
#ifdef WIN_DEBUG
												if (verts)
													res[95] = 10000000 + (DWORD)verts;
#endif
												//verts = NULL;
												if (!verts) {
													//verts = verts;
													verts = &vertsTemp;
												}
												if (verts) {
													_Verts(verts);
													trans_last = trans;
													verts->v.set(p);
													verts->trans = trans;
													verts->n_r.set(v->n_r);
													verts->obj = obj;
													if (verts != &vertsTemp)
														raytracing_verts.insertLink(&raytracing_verts, verts, NULL, NULL);
													__image = &verts->color;
													//__image = _raytracing;
													//*__image = RED;

													//shadow test set color to black or white
													//then stop ray tracing
													if (3 == ray.type) {
														*__image = Light3D_multi(ray.color, ray.f / 5);
														verts->type = 0;
														break;
													}
													else {
														n0.set(p);
														n1.set(n0)* cam->M_1;
														//get texture and normal vector at the same time
														*__image = obj->getTextureColor(obj, n0, n1, n2, n3, v, &verts->v_n);

														if (1 == obj->normal_type) {
															//get line formula
															//v0-v1
															Vert3D::GetLine(v1->v_s, v0->v_s, l1);
															//v1-v
															Vert3D::GetLine(v->v_s, v1->v_s, l);
															//v-v0
															Vert3D::GetLine(v0->v_s, v->v_s, l0);
															//get range x
															_n1.set(n0);
															cam->normalize(cam, _n1);
															_n2.set(_n1.x * cam->scale_w + cam->offset_w, _n1.y * cam->scale_h + cam->offset_h, _n1.z);
															EFTYPE __y = _n2.y;
															EFTYPE __x = _n2.x;
															INT _line_l1 = (INT)(l1.x * __y + l1.y);
															INT _line_l = (INT)(l.x * __y + l.y);
															INT _line_l0 = (INT)(l0.x * __y + l0.y);
															INT line_l, line_r;
															INT minx, maxx;
															minx = min(min(v->x0, v0->x0), min(v->x0, v1->x0));
															maxx = max(max(v->x0, v0->x0), max(v->x0, v1->x0));
															if (_line_l1 < minx || _line_l1 > maxx) {
																_line_l1 = 0;
																line_l = min(_line_l, _line_l0);
																line_r = max(_line_l, _line_l0);
															}
															else if (_line_l < minx || _line_l > maxx) {
																_line_l = 0;
																line_l = min(_line_l1, _line_l0);
																line_r = max(_line_l1, _line_l0);
															}
															else if (_line_l0 < minx || _line_l0 > maxx) {
																_line_l0 = 0;
																line_l = min(_line_l1, _line_l);
																line_r = max(_line_l1, _line_l);
															}
															else {
																line_l = min(min(_line_l, _line_l0), min(_line_l1, _line_l0));
																line_r = max(max(_line_l, _line_l0), max(_line_l1, _line_l0));
															}
															//get interpolation normal vector from 3 points of a triangle
															Object3D_GetInterpolationNormalVector(v0, v1, v, __x, __y,
																line_r, line_l, _line_l1, _line_l, _line_l0,
																5, _n0, _n1, _n2, _n3);
															verts->v_3.set(_n0);
														}
														else {
															_n0.set(v->n_r);
															verts->v_3.set(verts->v_n);
														}

														//calculate sumption of light factors
														lgt = man.lgts.link;
														f = 0;
														if (lgt) {
															do {
																f += lgt->getFactor(lgt, _n0, n0);

																if (device->render_light < 0) {
																	break;
																}

																lgt = man.lgts.next(&man.lgts, lgt);
															} while (lgt && lgt != man.lgts.link);
														}


														//normal verts
														if (0 == render_state) {
															*__image = Light3D_multi(*__image, f);
															//set type normal
															verts->type = 0;
														}
														//reflection verts
														else if (1 == render_state) {
															*__image = Light3D_add(*__image, EP_BLACK, f / 2);
															//*__image = Light3D::multi(EP_BLACK, f);
															//set type reflection
															verts->type = 1;
														}
														//transparent verts
														else if (2 == render_state) {
															*__image = Light3D_add(*__image, EP_BLACK, f / 2);
															//*__image = Light3D::multi(EP_BLACK, f);
															//set type transparent
															verts->type = 2;
														}
													}
												}
												//when the ray is reflection,
												//there will be one or two hit point
												//in other case, because of using backface cull,
												//there will be only one hit point
												if (!(1 == ray.type)) {
													break;
												}
											}
										}

										if (obj->vertex_type == 1) {
											v0 = NULL;
											v1 = NULL;
										}
										else if (obj->vertex_type == 2) {
											if ((traverseCount + 1) % 2 == 0) {
												v0 = v;
												//this->v1 = this->v1;
											}
											else {
												v0 = v1;
												v1 = v;
											}
										}
										else {
											v0 = v1;
											v1 = v;
										}
									}
									else if (v0 == NULL) {
										v0 = v;
									}
									else if (v1 == NULL) {
										v1 = v;
									}

									v = link->next(link, v);
								} while (v && v != link->link);
							}
						}

						// use the objects around or all the objects?
						if (&man.objs == olink) {
							//first do objects till end
							//then do reflection and then transparent object
							if (render_state == 0) {
								obj = man.objs.next(&man.objs, obj);
								if (!(obj && obj != man.objs.link)) {
									obj = man.refl.link;
									//next render reflection points
									render_state = 1;
									if (!obj) {
										//or render reflection points
										obj = man.tras.link;
										render_state = 2;
									}
								}
							}
							else if (render_state == 1) {
								obj = man.refl.next(&man.refl, obj);
								if (!(obj && obj != man.refl.link)) {
									obj = man.tras.link;
									//next render transparent points
									render_state = 2;
								}

							}
							else {
								obj = man.tras.next(&man.tras, obj);
								if (!(obj && obj != man.tras.link)) {
									break;
								}
							}
						}
						else {
							obj = olink->next(olink, obj);
							if (!(obj && obj != olink->link)) {
								break;
							}
						}
					} while (obj);
				}
				//make sure all temporary links are cleaned
				if (&man.objs == olink) {
				}
				else {
					olink->clearLink(olink);
				}
				//get the nearest verts from all the ray traced verts
				Verts * verts = raytracing_verts.link;
				Verts * nearest_vert = verts;
				if (verts) {
					do {
						if (verts->trans < nearest_vert->trans) {
							nearest_vert = verts;
						}

						verts = raytracing_verts.next(&raytracing_verts, verts);
					} while (verts && verts != raytracing_verts.link);
				}
				if (nearest_vert) {
					raytracing_verts_accumulated.insertLink(&raytracing_verts_accumulated, nearest_vert, NULL, NULL);
					//RAYTRACING_MUTEX(raytracing_verts.~MultiLinkList(););
					raytracing_verts.clearLink(&raytracing_verts);

					//normal verts
					if (0 == nearest_vert->type) {
						//get shadow test ray
						cur_lgt = man.lgts.link;
						if (cur_lgt) {
							do {
								n2.set(0, 0, 0, 1) * cur_lgt->M *cam->M;
								n2 - nearest_vert->v;
								n2.normalize();
								ray.set(nearest_vert->v, n2);
								//set object
								ray.obj = nearest_vert->obj;
								//set ray type
								ray.type = 3;
								//this color
								ray.color = nearest_vert->color;
								//this factor
								ray.f = cur_lgt->getFactor(cur_lgt, nearest_vert->v_3, nearest_vert->v);

								__image = &nearest_vert->color;

								*__image = ray.color;
								Obj3D * obj = olink->link;
								if (obj) {
									int render_state = 0;
									VObj * v, *v0, *v1, *vtemp;

									EFTYPE trans_last = 1000;
									// for each triangle
									do {
										//object aabb intersection
										INT intersection = 1;
										if (&man.objs == olink) {
											//intersection = man.octTree.Collision(&man.octTree, man.octTree.link, ray.original, ray.direction, (Camera3D*)cam, obj);
											intersection = Collision(ray.original, ray.direction, obj, trans_last);
										}
										if (intersection) {

											// when the ray is reflection
											// then use all the verts instead 
											// of the verts after frustrum culling
											if (1 == ray.type) {
												link = &obj->verts;
											}
											else {
												link = &obj->verts_r;
											}
											v = link->link;
											// more than 3 verts
											if (v && link->linkcount >= 3) {
												v0 = NULL; v1 = NULL;
												int traverseCount = 0;
												do {
													traverseCount++;
													//there must be three verts
													if (v0 && v1) {
														// back face culling
														// when the ray is reflection or shadow testing
														// then do not need back face culling
														if (v->backface > 0 || 1 == ray.type || (3 == ray.type && obj->backfaceculling == 0))
														{
															//NOTE: ray tracing is in camera coordinate
															//get intersect point
															trans = Vert3D::GetLineIntersectPointWithTriangle(v->v_c, v0->v_c, v1->v_c, ray.original, ray.direction, trans_last, p);
															//trans is greater than zero, and less than last trans
															if (EP_GTZERO(trans)) {
																trans_last = trans;
																*__image = Light3D_multi(*__image, ray.f / 5);

																//caustic affect on refraction
																if (render_state == 2) {
																	n0.set(p);
																	n1.set(n0)* cam->M_1;
																	//get texture and normal vector at the same time
																	obj->getTextureColor(obj, n0, n1, n2, n3, v, NULL);

																	if (1 == obj->normal_type) {
																		//get line formula
																		//v0-v1
																		Vert3D::GetLine(v1->v_s, v0->v_s, l1);
																		//v1-v
																		Vert3D::GetLine(v->v_s, v1->v_s, l);
																		//v-v0
																		Vert3D::GetLine(v0->v_s, v->v_s, l0);
																		//get range x
																		_n1.set(n0);
																		cam->normalize(cam, _n1);
																		_n2.set(_n1.x * cam->scale_w + cam->offset_w, _n1.y * cam->scale_h + cam->offset_h, _n1.z);
																		EFTYPE __y = _n2.y;
																		EFTYPE __x = _n2.x;
																		INT _line_l1 = (INT)(l1.x * __y + l1.y);
																		INT _line_l = (INT)(l.x * __y + l.y);
																		INT _line_l0 = (INT)(l0.x * __y + l0.y);
																		INT line_l, line_r;
																		INT minx, maxx;
																		minx = min(min(v->x0, v0->x0), min(v->x0, v1->x0));
																		maxx = max(max(v->x0, v0->x0), max(v->x0, v1->x0));
																		if (_line_l1 < minx || _line_l1 > maxx) {
																			_line_l1 = 0;
																			line_l = min(_line_l, _line_l0);
																			line_r = max(_line_l, _line_l0);
																		}
																		else if (_line_l < minx || _line_l > maxx) {
																			_line_l = 0;
																			line_l = min(_line_l1, _line_l0);
																			line_r = max(_line_l1, _line_l0);
																		}
																		else if (_line_l0 < minx || _line_l0 > maxx) {
																			_line_l0 = 0;
																			line_l = min(_line_l1, _line_l);
																			line_r = max(_line_l1, _line_l);
																		}
																		else {
																			line_l = min(min(_line_l, _line_l0), min(_line_l1, _line_l0));
																			line_r = max(max(_line_l, _line_l0), max(_line_l1, _line_l0));
																		}
																		//get interpolation normal vector from 3 points of a triangle
																		Object3D_GetInterpolationNormalVector(v0, v1, v, __x, __y,
																			line_r, line_l, _line_l1, _line_l, _line_l0,
																			5, _n0, _n1, _n2, _n3);
																	}
																	else {
																		_n0.set(v->n_r);
																	}
																	//get refraction ray
																	// refraction vector
																	//T = ((nL / nT) * N * L - SQRT(1 - (nL^2 / nT ^2)*[1 - (N * L)^2])) * N - (nL / nT) * L
																	//get n3 = N
																	//n3.set(nearest_vert->v_n);
																	n3.set(_n0);// .negative();
																				//get n2 = L
																				//this formula used a negative I
																	n2.set(ray.direction);// .negative();
																						  //get n3 = T
																	EFTYPE cross = n2 ^ n3;
																	//sin(oL) <= nT / nL, that is nT > nL
																	EFTYPE nL = 0.1, nT = 0.5;
																	EFTYPE nL_nT = nL / nT;
																	EFTYPE pN = nL_nT * cross - sqrt(1 - nL_nT * nL_nT * (1 - cross * cross));
																	n3 * pN;
																	n2 * nL_nT;
																	n3 - n2;
																	n3.normalize();

																	EFTYPE f = cur_lgt->getFactor(cur_lgt, n3, p);
																	*__image = Light3D_multi(*__image, f * 5);
																}
																break;
															}
														}

														if (obj->vertex_type == 1) {
															v0 = NULL;
															v1 = NULL;
														}
														else if (obj->vertex_type == 2) {
															if ((traverseCount + 1) % 2 == 0) {
																v0 = v;
																//this->v1 = this->v1;
															}
															else {
																v0 = v1;
																v1 = v;
															}
														}
														else {
															v0 = v1;
															v1 = v;
														}
													}
													else if (v0 == NULL) {
														v0 = v;
													}
													else if (v1 == NULL) {
														v1 = v;
													}

													v = link->next(link, v);
												} while (v && v != link->link);
											}
										}

										// use the objects around or all the objects?
										if (&man.objs == olink) {
											//first do objects till end
											//then do reflection and then transparent object
											if (render_state == 0) {
												obj = man.objs.next(&man.objs, obj);
												if (!(obj && obj != man.objs.link)) {
													obj = man.refl.link;
													//next render reflection points
													render_state = 1;
													if (!obj) {
														//or render reflection points
														obj = man.tras.link;
														render_state = 2;
													}
												}
											}
											else if (render_state == 1) {
												obj = man.refl.next(&man.refl, obj);
												if (!(obj && obj != man.refl.link)) {
													obj = man.tras.link;
													//next render transparent points
													render_state = 2;
												}

											}
											else {
												obj = man.tras.next(&man.tras, obj);
												if (!(obj && obj != man.tras.link)) {
													break;
												}
											}
										}
										else {
											obj = olink->next(olink, obj);
											if (!(obj && obj != olink->link)) {
												break;
											}
										}
									} while (obj);
								}
								if (device->render_light < 0) {
									break;
								}

								cur_lgt = man.lgts.next(&man.lgts, cur_lgt);
							} while (cur_lgt && cur_lgt != man.lgts.link);
						}
						break;
						/*
						if (cur_lgt && (shadow_count == 0 || cur_lgt != man.lgts.link)) {
						n2.set(0, 0, 0, 1) * cur_lgt->M *cam->M;
						n2 - nearest_vert->v;
						n2.normalize();
						ray.set(nearest_vert->v, n2);
						//set object
						ray.obj = nearest_vert->obj;
						//set ray type
						ray.type = 3;
						//this color
						ray.color = nearest_vert->color;
						//this factor
						ray.f = cur_lgt->getFactor(nearest_vert->v_3, nearest_vert->v);
						//test same direction
						//EFTYPE cross = n2 & nearest_vert->v_3;
						//if (cross < 0) {
						//	//not same direction, this vertex is in shadow
						//	nearest_vert->color = Light3D_multi(nearest_vert->color, ray.f / 2);
						//}

						//shadow test does not affect ray tracing times
						count++;

						//get next shadow test light
						shadow_count++;
						nearest_vert_shadow = nearest_vert;
						cur_lgt = man.lgts.next(&man.lgts, cur_lgt);
						}
						else {
						//stop ray tracing
						break;
						}
						*/
					}
					//reflection verts
					else if (1 == nearest_vert->type) {
						//get reflection ray
						// reflection vector
						// R = I -  N * ( dot(I , N)* 2 )
						//get n3 = N
						//n3.set(nearest_vert->v_n);
						n3.set(nearest_vert->v_3);
						//get n2 = I
						n2.set(ray.direction);
						//get n2 = R
						EFTYPE cross = n2 ^ n3;
						n3 * (cross * 2);
						n2 - n3;
						n2.normalize();// .negative();
									   //set ray
						ray.set(nearest_vert->v, n2);
						//set object
						ray.obj = nearest_vert->obj;
						//set ray type
						ray.type = 1;
					}
					//transparent verts
					else if (2 == nearest_vert->type) {
						//get refraction ray
						// refraction vector
						//T = ((nL / nT) * N * L - SQRT(1 - (nL^2 / nT ^2)*[1 - (N * L)^2])) * N - (nL / nT) * L
						//get n3 = N
						//n3.set(nearest_vert->v_n);
						n3.set(nearest_vert->v_3);
						//get n2 = L
						//this formula used a negative I
						n2.set(ray.direction).negative();
						//get n3 = T
						EFTYPE cross = n2 ^ n3;
						//sin(oL) <= nT / nL, that is nT > nL
						EFTYPE nL = 0.1, nT = 0.5;
						EFTYPE nL_nT = nL / nT;
						EFTYPE pN = nL_nT * cross - sqrt(1 - nL_nT * nL_nT * (1 - cross * cross));
						n3 * pN;
						n2 * nL_nT;
						n3 - n2;
						n3.normalize();
						//set ray
						ray.set(nearest_vert->v, n3);
						//set object
						ray.obj = nearest_vert->obj;
						//set ray type
						ray.type = 2;
					}
				}
				else {
					break;
				}

			} while (--count > 0);

			//RAYTRACING_MUTEX(raytracing_verts.~MultiLinkList(););
			raytracing_verts.clearLink(&raytracing_verts);

			//accumulate all the ray traced verts' color
			Verts * verts = raytracing_verts_accumulated.link;
			DWORD color = EP_BLACK;
			if (verts) {
				do {
					//if (0 == verts->type) 
					{
						color = Light3D_add(color, verts->color, 0.1);
					}

					verts = raytracing_verts_accumulated.next(&raytracing_verts_accumulated, verts);
				} while (verts && verts != raytracing_verts_accumulated.link);
			}
			//RAYTRACING_MUTEX(raytracing_verts_accumulated.~MultiLinkList(););
			raytracing_verts_accumulated.clearLink(&raytracing_verts_accumulated);

			*_raytracing = color;
		}
	}
#endif
}

__global__ void initializeKernel(EFTYPE * res, INT size, Manager3D * _man, VObjPoolImp * vobjPoolImp, ObjPoolImp * objPoolImp, CamPoolImp *camPoolImp, LgtPoolImp *lgtPoolImp, TexturePoolImp *textPoolImp, TextureManager *_tman , OctPoolImp * octPoolImp, Group3DPoolImp * group3DPoolImp)
{
#ifdef RUN_DEVICE
	int index = 0;

	TextureManager& tman = *_tman;
	tman.texturePoolImp = textPoolImp;
	tman.Init();
	tman.Reload();

	Manager3D& man = *_man;
	Obj3D * obj = NULL;

	man.vobjPoolImp = vobjPoolImp;
	man.objPoolImp = objPoolImp;
	man.camPoolImp = camPoolImp;
	man.lgtPoolImp = lgtPoolImp;
	man.octPoolImp = octPoolImp;
	man.group3DPoolImp = group3DPoolImp;
	_VObjPoolImp(vobjPoolImp);
	_ObjPoolImp(objPoolImp, vobjPoolImp);
	_CamPoolImp(camPoolImp);
	_LgtPoolImp(lgtPoolImp);
	_VObjPoolImp(vobjPoolImp);
	_OctPoolImp(octPoolImp, objPoolImp, vobjPoolImp);
	_Group3DPoolImp(group3DPoolImp, objPoolImp, vobjPoolImp);
	man.Init();
	Camera3D * cam = &man.addCamera(50, 50, 50, 1000, 70, 70);
	man.setCameraRange(500, 240, 126, 126);
	cam->_move(cam, 0, 0, -200);

	man.addLight(9, 100, 300);

	Object3D *_obj = &man.addObject();
	_obj->addVert(_obj, -10, -10, 10).addVert(_obj, 10, -10, 10).addVert(_obj, -10, 10, 10).addVertA(_obj, 10, 10, 10, -1)._scale(_obj, 5, 5, 5)
		._move(_obj, 0, 100, -200).setColor(_obj, GREEN).setTexture(_obj, tman, 1, 0).setUV(_obj, 0, 0);
	_obj = &man.addObject();
	_obj->addVert(_obj, -10,0,-10).addVert(_obj, 10, 0, -10).addVert(_obj, -10, 0, 10).addVertA(_obj, 10, 0, 10, -1)._rotate(_obj, 0, 0, 180)
		._scale(_obj, 5, 5, 5)._move(_obj, 250, -40, 250).setColor(_obj, LIGHTGRAY);

	_obj = &man.addObject();
	_obj->addVert(_obj, -10, -10, 10).addVert(_obj, 10, -10, 10).addVert(_obj, -10, 10, 10).addVertA(_obj, 10, 10, 10, -1)._scale(_obj, 5, 5, 5)
		._move(_obj, 0, 100, -100).setColor(_obj, GREEN).setTexture(_obj, tman, 1, 0).setUV(_obj, 0, 0);

	_obj = &man.addObject();
	_obj->addVert(_obj, -10, -10, 10).addVert(_obj, 10, -10, 10).addVert(_obj, -10, 10, 10).addVertA(_obj, 10, 10, 10, -1)._scale(_obj, 5, 5, 5)
		._move(_obj, 0, 100, -0).setColor(_obj, GREEN).setTexture(_obj, tman, 1, 0).setUV(_obj, 0, 0);

	_obj = &man.addObject();
	_obj->addVert(_obj, -10, -10, 10).addVert(_obj, 10, -10, 10).addVert(_obj, -10, 10, 10).addVertA(_obj, 10, 10, 10, -1)._scale(_obj, 5, 5, 5)
		._move(_obj, 0, 100, 100).setColor(_obj, GREEN).setTexture(_obj, tman, 1, 0).setUV(_obj, 0, 0);

	man.initialized = 1;


#ifdef WIN_DEBUG
	
	if (cam) {
		//res[index++] = (DWORD)cam;
		//res[index++] = (DWORD)&man.camTemp;

		res[index++] = cam->M.mx.x + cam->M.mx.y + cam->M.mx.z + cam->M.mx.w + cam->M.my.x + cam->M.my.y + cam->M.my.z + cam->M.my.w;
		res[index++] = cam->_M.Ms.mx.x + cam->_M.Ms.mx.y + cam->_M.Ms.mx.z + cam->_M.Ms.mx.w + cam->_M.Ms.my.x + cam->_M.Ms.my.y + cam->_M.Ms.my.z + cam->_M.Ms.my.w;
		res[index++] = cam->_M.M->mx.x + cam->_M.M->mx.y + cam->_M.M->mx.z + cam->_M.M->mx.w + cam->_M.M->my.x + cam->_M.M->my.y + cam->_M.M->my.z + cam->_M.M->my.w;
		res[index++] = cam->_M.Mm.mx.x + cam->_M.Mm.mx.y + cam->_M.Mm.mx.z + cam->_M.Mm.mx.w + cam->_M.Mm.my.x + cam->_M.Mm.my.y + cam->_M.Mm.my.z + cam->_M.Mm.my.w;
		res[index++] = cam->_M.Mrx.mx.x + cam->_M.Mrx.mx.y + cam->_M.Mrx.mx.z + cam->_M.Mrx.mx.w + cam->_M.Mrx.my.x + cam->_M.Mrx.my.y + cam->_M.Mrx.my.z + cam->_M.Mrx.my.w;
	}

	obj = (Obj3D*)_obj;
	if (obj) {
		res[index++] = obj->M.mx.x + obj->M.mx.y + obj->M.mx.z + obj->M.mx.w + obj->M.my.x + obj->M.my.y + obj->M.my.z + obj->M.my.w;
		res[index++] = obj->_M.Ms.mx.x + obj->_M.Ms.mx.y + obj->_M.Ms.mx.z + obj->_M.Ms.mx.w + obj->_M.Ms.my.x + obj->_M.Ms.my.y + obj->_M.Ms.my.z + obj->_M.Ms.my.w;
		res[index++] = obj->_M.M->mx.x + obj->_M.M->mx.y + obj->_M.M->mx.z + obj->_M.M->mx.w + obj->_M.M->my.x + obj->_M.M->my.y + obj->_M.M->my.z + obj->_M.M->my.w;
		res[index++] = obj->_M.Mm.mx.x + obj->_M.Mm.mx.y + obj->_M.Mm.mx.z + obj->_M.Mm.mx.w + obj->_M.Mm.my.x + obj->_M.Mm.my.y + obj->_M.Mm.my.z + obj->_M.Mm.my.w;
		res[index++] = obj->_M.Mrx.mx.x + obj->_M.Mrx.mx.y + obj->_M.Mrx.mx.z + obj->_M.Mrx.mx.w + obj->_M.Mrx.my.x + obj->_M.Mrx.my.y + obj->_M.Mrx.my.z + obj->_M.Mrx.my.w;
	}

	res[index++] = 111111;
	res[index++] = (DWORD)&man;
	res[index++] = man.initialized;
	res[index++] = man.objs.linkcount;
	res[index++] = (DWORD)man.objs.link;
	res[index++] = 111111;

	obj = man.objs.link;
	if (obj) {
		do {

			VObj * v = obj->verts.link;
			res[index++] = (DWORD)obj;
			res[index++] = (DWORD)obj->verts.link;
			if (v) {
				do {

					res[index++] = v->v.x;
					res[index++] = v->v.y;
					res[index++] = v->v.z;
					res[index++] = v->v.w;

					v = obj->verts.next(&obj->verts, v);
				} while (v && v != obj->verts.link);
			}

			obj = man.objs.next(&man.objs, obj);
		} while (obj && obj != man.objs.link);
	}
#endif
#endif
}

// Helper function for using CUDA
hipError_t initializeWithCuda(EFTYPE * res, int res_size, Manager3D * man)
{
	hipError_t cudaStatus;

	DWORD** threadImage = NULL;
	TextureLocalManager tman;
	TextureLocalPoolImp textPool;
	_TextureLocalPoolImp(&textPool);
	tman.texturePoolImp = &textPool;
	tman.Init();
	INT tid = tman.addTexture(64, 64, 8);
	fprintf(stderr, "added texture: %d", tid);
	tid = tman.addTexture("image/1.jpg");
	fprintf(stderr, "added texture: %d", tid);

	// Allocate GPU buffers of objects
	// alloc golbal symbols
	///////////////////////////////////////
	//int *tmp;
	//hipMalloc((void **)&tmp, sizeof(int)* 10);
	//hipMemcpyToSymbol(HIP_SYMBOL(a), &tmp, sizeof(int *), size_t(0), hipMemcpyHostToDevice);
	//Manager3D * dev_man_temp;

	cudaStatus = hipMalloc(&octPoolImp, sizeof(OctPoolImp));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc OctPoolImp failed!");
		goto Error;
	}

	Group3DPoolImp * group3DPoolImp;
	cudaStatus = hipMalloc(&group3DPoolImp, sizeof(Group3DPoolImp));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc Group3DPoolImp failed!");
		goto Error2_1;
	}

	//int verts_pool_count = 0;
	//VertsPoolImp ** _verts_pool = new VertsPoolImp*[THREAD_W * THREAD_H];
	//for (int i = 0; i < THREAD_W * THREAD_H; i++) {
	//	cudaStatus = hipMalloc((void**)&_verts_pool[i], sizeof(VertsPoolImp));
	//	if (cudaStatus != hipSuccess) {
	//		fprintf(stderr, "hipMalloc failed!%s\n", hipGetErrorString(cudaStatus));
	//		goto Error_1;
	//	}
	//	verts_pool_count ++;
	//}
	//cudaStatus = hipMalloc((void**)&verts_pool, sizeof(VertsPoolImp*) * THREAD_W * THREAD_H);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!%s\n", hipGetErrorString(cudaStatus));
	//	goto Error_1;
	//}
	//cudaStatus = hipMemcpy(verts_pool, _verts_pool, sizeof(VertsPoolImp*) * THREAD_W * THREAD_H, hipMemcpyHostToDevice);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed for verts_pool: %s", hipGetErrorString(cudaStatus));
	//	goto Error_1;
	//}
	cudaStatus = hipMalloc((void**)&verts_pool, sizeof(VertsPoolImp) * THREAD_W * THREAD_H);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for verts_pool: %s", hipGetErrorString(cudaStatus));
		goto Error_1;
	}

	cudaStatus = hipMalloc((void**)&dev_man, sizeof(Manager3D));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(dev_man), &dev_man_temp, sizeof(dev_man_temp), size_t(0), hipMemcpyHostToDevice);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpyToSymbol failed!");
	//	goto Error;
	//}

	hipMemset((void*)&dev_man->initialized, 0, 4);//sizeof(int) = 4 * sizeof(byte)
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error1;
	}
	
	//Texture
	cudaStatus = hipMalloc((void**)&dev_tman, sizeof(TextureManager));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for texture!");
		goto Error1;
	}
	TexturePoolImp *textPoolImp;
	cudaStatus = hipMalloc(&textPoolImp, sizeof(TexturePoolImp));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for texture poll!");
		goto Error1_1;
	}
	//verts
	cudaStatus = hipMalloc((void**)&dev_vman, sizeof(VertsMan*) * THREAD_W * THREAD_H * 2);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for verts!");
		goto Error1_2;
	}
	int vman_count = 0;
	VertsMan** vman = new VertsMan*[THREAD_W * THREAD_H * 2];
	for (int i = 0; i < THREAD_W * THREAD_H * 2; i++) {
		cudaStatus = hipMalloc((void**)&vman[i], sizeof(VertsMan));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed for verts!");
			goto Error1_3;
		}
		vman_count++;
	}
	cudaStatus = hipMemcpy(dev_vman, vman, sizeof(VertsMan*) * THREAD_W * THREAD_H * 2, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for verts (%s)!", hipGetErrorString(cudaStatus));
		goto Error1_3;
	}

	//make pool copy
	TextureLocalPoolImp textPoolBackup;
	//memcpy(&textPoolBackup, &textPool, sizeof(TextureLocalPoolImp));
	for (int i = 0; i < MAX_TEXTURELOCAL; i++) {
		textPoolBackup.pool[i].texture = NULL;
	}
	//texture copy
	for (int i = 0; i < MAX_TEXTURELOCAL; i++) {
		TextureLocal *textLocal = &textPool.pool[i];
		if (!textLocal) {
			continue;
		}
		if (textLocal->texture != NULL && textLocal->width > 0 && textLocal->height > 0) {
			fprintf(stderr, "copying texture %d: width: %d, height: %d, p: %p", i, textLocal->width, textLocal->height, textLocal->texture);
			DWORD * texture;
			cudaStatus = hipMalloc(&texture, sizeof(DWORD)* textLocal->width * textLocal->height);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed for texture %d!", i);
				break;
			}
			fprintf(stderr, "got dev p: %p", texture);
			cudaStatus = hipMemcpy(texture, textLocal->texture,  sizeof(DWORD)* textLocal->width * textLocal->height, hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed for texture p: %p", texture);
				break;
			}
			//change texture pointer
			textPoolBackup.pool[i].texture = texture;
			textPoolBackup.pool[i].width = textLocal->width;
			textPoolBackup.pool[i].height = textLocal->height;
			textPoolBackup.pool[i].uniqueID = textLocal->uniqueID;
		}

	}
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for texture!");
		goto Error1_5;
	}
	//make suer TexturePoolImp and TextureLocalPoolImp are same size
	if (sizeof(TexturePoolImp) != sizeof(TextureLocalPoolImp)) {
		fprintf(stderr, "TexturePoolImp and TextureLocalPoolImp are not the same size!");
		goto Error1_5;
	}
	//texture pool copy
	cudaStatus = hipMemcpy(textPoolImp, &textPoolBackup, sizeof(TexturePoolImp), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for texture pool!");
		goto Error1_5;
	}


	VObjPoolImp *vobjPoolImp;
	cudaStatus = hipMalloc(&vobjPoolImp, sizeof(VObjPoolImp));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error2;
	}
	ObjPoolImp *objPoolImp;
	cudaStatus = hipMalloc(&objPoolImp, sizeof(ObjPoolImp));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error3;
	}
	CamPoolImp *camPoolImp;
	cudaStatus = hipMalloc(&camPoolImp, sizeof(CamPoolImp));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error4;
	}
	LgtPoolImp *lgtPoolImp;
	cudaStatus = hipMalloc(&lgtPoolImp, sizeof(LgtPoolImp));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error5;
	}
	cudaStatus = hipMalloc((void**)&dev_res, sizeof(DWORD)* res_size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error6;
	}
	cudaStatus = hipMalloc((void**)&dev_resf, sizeof(EFTYPE)* res_size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error7;
	}
	cudaStatus = hipMalloc((void**)&objIterator, sizeof(Obj3D*)* MAX_ITERATOR);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error8;
	}
	cudaStatus = hipMalloc((void**)&tgIterator, sizeof(Triangles)* MAX_ITERATOR);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error9;
	}
	cudaStatus = hipMalloc((void**)&_device, sizeof(Device));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error10;
	}
	device.Init(WIN_WIDTH, WIN_HEIGHT);
	for (int i = 0; i < 2; i++) {
		cudaStatus = hipMalloc((void**)&device.float_a[i], sizeof(EFTYPE)* device.width * device.height);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error11;
		}
	}
	for (int i = 0; i < 3; i++) {
		cudaStatus = hipMalloc((void**)&device.dword_a[i], sizeof(DWORD)* device.width * device.height);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error12;
		}
	}
	cudaStatus = hipMalloc((void**)&device.threadImage, sizeof(DWORD*)* MAX_ITERATOR);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error13;
	}
	threadImage = new DWORD*[MAX_ITERATOR];
	device.threadImageCount = 0;
	for (int i = 0; i < MAX_ITERATOR; i++) {
		cudaStatus = hipMalloc((void**)&threadImage[i], sizeof(DWORD)* device.width * device.height);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error14;
		}
		device.threadImageCount++;
	}
	cudaStatus = hipMemcpy(device.threadImage, threadImage, sizeof(DWORD*)* MAX_ITERATOR, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error14;
	}
	cudaStatus = hipMemcpy(_device, &device, sizeof(Device), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error14;
	}
	tango = device.tango;
	depth = device.depth;

	cudaStatus = hipMemset(dev_resf, 0, sizeof(EFTYPE)* res_size);
	if (cudaStatus != cudaStatus) {
		fprintf(stderr, "hipMemset failed!");
		goto Error14;
	}
	///////////////////////////////////////

	//�˺���������
	//<<<�鲢�������̲߳�������ÿ����ʹ�õĹ����ڴ��С��������>>>
	// Launch a kernel on the GPU with one thread for each element.
	initializeKernel << <1, 1 >> > (dev_resf, res_size, dev_man, vobjPoolImp, objPoolImp, camPoolImp, lgtPoolImp, textPoolImp, dev_tman, octPoolImp, group3DPoolImp);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "initializeKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error14;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching initializeKernel!\n", cudaStatus);
		fprintf(stderr, "initializeKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error14;
	}

#ifdef WIN_DEBUG
	///////////////////////////////////////
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(res, dev_resf, res_size * sizeof(EFTYPE), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error14;
	}
	///////////////////////////////////////
	DEBUG("��ʼ��:");
	for (int i = 0; i < MAX_ITERATOR; i++) {
		DEBUG("%.2f->", res[i]);
	}
	DEBUG("\n");
#endif

	goto Error;
	///////////////////////////////////////
Error14:
	for (int i = 0; threadImage && i < device.threadImageCount && i < MAX_ITERATOR; i++) {
		if (threadImage[i]) {
			hipFree(threadImage[i]);
		}
	}
Error13:
	hipFree(device.threadImage);
Error12 :
	for (int i = 0; i < 3; i++) {
		if (device.dword_a[i]) {
			hipFree(device.dword_a[i]);
		}
	}
Error11 :
	for (int i = 0; i < 2; i++) {
		if (device.float_a[i]) {
			hipFree(device.float_a[i]);
		}
	}
	hipFree(_device);
Error10:
	hipFree(tgIterator);
Error9:
	hipFree(objIterator);
Error8:
	hipFree(dev_resf);
Error7:
	hipFree(dev_res);
Error6:
	hipFree(lgtPoolImp);
Error5:
	hipFree(camPoolImp);
Error4:
	hipFree(objPoolImp);
Error3:
	hipFree(vobjPoolImp);
Error2:
	//hipFree(dev_res);
	hipFree(textPoolImp);
Error1_5:
	for (int i = 0; i < MAX_TEXTURELOCAL; i++) {
		if (textPoolBackup.pool[i].texture != NULL) {
			hipFree(textPoolBackup.pool[i].texture);
		}
	}
Error1_3:
	for (int i = 0; i < vman_count && i < THREAD_W * THREAD_H; i++) {
		hipFree(vman[i]);
	}
Error1_2:
	hipFree(dev_vman);
Error1_1:
	hipFree(dev_tman);
Error1:
	hipFree(dev_man);
Error_1:
	//for (int i = 0; _verts_pool && i < verts_pool_count && i < THREAD_W * THREAD_H; i++) {
	//	if (_verts_pool[i]) {
	//		hipFree(_verts_pool[i]);
	//	}
	//}
	if (verts_pool) {
		//delete[] verts_pool;
		//verts_pool = NULL;
		hipFree(verts_pool);
	}
Error2_1:
	//hipFree(octPoolImp);
	///////////////////////////////////////
Error:
	  if (threadImage) {

		  delete[] threadImage;
		  threadImage = NULL;
	  }
	return cudaStatus;
}

// Helper function for using CUDA
hipError_t unInitializeWithCuda(EFTYPE * res, int res_size, Manager3D * man)
{
	hipError_t cudaStatus = hipSuccess;

	///////////////////////////////////////
Error14:
	//for (int i = 0; threadImage && i < device.threadImageCount && i < THREAD_W * THREAD_H; i++) {
	//	if (threadImage[i]) {
	//		hipFree(threadImage[i]);
	//	}
	//}
Error13:
	hipFree(device.threadImage);
Error12:
	for (int i = 0; i < 6; i++) {
		if (device.dword_a[i]) {
			hipFree(device.dword_a[i]);
		}
	}
Error11:
	for (int i = 0; i < 3; i++) {
		if (device.float_a[i]) {
			hipFree(device.float_a[i]);
		}
	}
	hipFree(_device);
Error10:
	hipFree(tgIterator);
Error9:
	hipFree(objIterator);
Error8:
	hipFree(dev_resf);
Error7:
	hipFree(dev_res);
	//Error6:
	//	hipFree(lgtPoolImp);
	//Error5:
	//	hipFree(camPoolImp);
	//Error4:
	//	hipFree(objPoolImp);
	//Error3:
	//	hipFree(vobjPoolImp);
	//Error2:
	//	//hipFree(dev_res);
	//	hipFree(textPoolImp);
	//Error1_5:
	//	for (int i = 0; i < MAX_TEXTURELOCAL; i++) {
	//		if (textPoolBackup.pool[i].texture != NULL) {
	//			hipFree(textPoolBackup.pool[i].texture);
	//		}
	//	}
Error1_1:
	hipFree(dev_tman);
Error1:
	hipFree(dev_man);
Error_1:
	if (verts_pool) {
		//delete[] verts_pool;
		//verts_pool = NULL;
		hipFree(verts_pool);
	}
	//Error2_1:
	//	hipFree(octPoolImp);
		///////////////////////////////////////
Error:
	//if (threadImage) {

	   // delete[] threadImage;
	   // threadImage = NULL;
	//}
	return cudaStatus;
}

__global__ void normalizeKernel(EFTYPE * res, INT size, Manager3D * _man, Obj3D ** objIterator, INT width, INT iteratorW, INT iteratorH)
{
#ifdef RUN_DEVICE
	int index = 0;

	Manager3D& man = *_man;
	Obj3D * obj = NULL;
	Cam3D * cam = NULL;

	INT iteratorIndex = 0;
	int iteratorIndexX = blockIdx.x * iteratorW;
	int iteratorIndexY = blockIdx.y * iteratorH;

	cam = man.cams.link;
	res[99] = (DWORD)cam;

	Obj3D * _obj = NULL;
	for (int i = iteratorIndexX; i < iteratorIndexX + iteratorW; i++) {
		for (int j = iteratorIndexY; j < iteratorIndexY + iteratorH; j++) {
			Obj3D * obj = objIterator[j * width + i];

			if (obj) {
				if (!_obj) _obj = obj;
				obj->render_normalize(obj, 2);
				res[50 + j * width + i] = obj->debugger;
			}
		}
	}

#ifdef WIN_DEBUG

	if (cam) {
		res[index++] = cam->M.mx.x + cam->M.mx.y + cam->M.mx.z + cam->M.mx.w + cam->M.my.x + cam->M.my.y + cam->M.my.z + cam->M.my.w;
		res[index++] = cam->_M.Ms.mx.x + cam->_M.Ms.mx.y + cam->_M.Ms.mx.z + cam->_M.Ms.mx.w + cam->_M.Ms.my.x + cam->_M.Ms.my.y + cam->_M.Ms.my.z + cam->_M.Ms.my.w;
		res[index++] = cam->_M.M->mx.x + cam->_M.M->mx.y + cam->_M.M->mx.z + cam->_M.M->mx.w + cam->_M.M->my.x + cam->_M.M->my.y + cam->_M.M->my.z + cam->_M.M->my.w;
		res[index++] = cam->_M.Mm.mx.x + cam->_M.Mm.mx.y + cam->_M.Mm.mx.z + cam->_M.Mm.mx.w + cam->_M.Mm.my.x + cam->_M.Mm.my.y + cam->_M.Mm.my.z + cam->_M.Mm.my.w;
		res[index++] = cam->_M.Mrx.mx.x + cam->_M.Mrx.mx.y + cam->_M.Mrx.mx.z + cam->_M.Mrx.mx.w + cam->_M.Mrx.my.x + cam->_M.Mrx.my.y + cam->_M.Mrx.my.z + cam->_M.Mrx.my.w;
	}

	obj = man.objs.link;
	if (obj) {
		res[index++] = obj->M.mx.x + obj->M.mx.y + obj->M.mx.z + obj->M.mx.w + obj->M.my.x + obj->M.my.y + obj->M.my.z + obj->M.my.w;
		res[index++] = obj->_M.Ms.mx.x + obj->_M.Ms.mx.y + obj->_M.Ms.mx.z + obj->_M.Ms.mx.w + obj->_M.Ms.my.x + obj->_M.Ms.my.y + obj->_M.Ms.my.z + obj->_M.Ms.my.w;
		res[index++] = obj->_M.M->mx.x + obj->_M.M->mx.y + obj->_M.M->mx.z + obj->_M.M->mx.w + obj->_M.M->my.x + obj->_M.M->my.y + obj->_M.M->my.z + obj->_M.M->my.w;
		res[index++] = obj->_M.Mm.mx.x + obj->_M.Mm.mx.y + obj->_M.Mm.mx.z + obj->_M.Mm.mx.w + obj->_M.Mm.my.x + obj->_M.Mm.my.y + obj->_M.Mm.my.z + obj->_M.Mm.my.w;
		res[index++] = obj->_M.Mrx.mx.x + obj->_M.Mrx.mx.y + obj->_M.Mrx.mx.z + obj->_M.Mrx.mx.w + obj->_M.Mrx.my.x + obj->_M.Mrx.my.y + obj->_M.Mrx.my.z + obj->_M.Mrx.my.w;
	}

	res[index++] = 1111111111111;
	res[index++] = blockDim.x;
	res[index++] = iteratorIndexX;
	res[index++] = iteratorIndexY;
	res[index++] = (DWORD)&man;
	res[index++] = man.initialized;
	res[index++] = man.objs.linkcount;
	res[index++] = (DWORD)man.objs.link;
	res[index++] = 1111111111111;

	obj = man.objs.link;
	if (obj) {
		do {

			VObj * v = obj->verts.link;
			if (v) {
				do {

					res[index++] = v->v_c.x;
					res[index++] = v->v_c.y;
					res[index++] = v->v_c.z;
					res[index++] = v->v_c.w;

					v = obj->verts.next(&obj->verts, v);
				} while (v && v != obj->verts.link);
			}

			obj = man.objs.next(&man.objs, obj);
		} while (obj && obj != man.objs.link);
	}
#endif
#endif
}

__global__ void rotateKernel(EFTYPE * res, INT size, Manager3D * _man, Obj3D ** objIterator, INT iteratorSize, EFTYPE ax, EFTYPE ay, EFTYPE az)
{
#ifdef RUN_DEVICE
	int index = 0;

	Manager3D& man = *_man;
	Obj3D * obj = NULL;
	Cam3D * cam = NULL;

	if (man.cams.link) {
		cam = man.cams.link;
		man.cams.link->_rotate(man.cams.link, ax, ay, az);
	}

	INT iteratorIndex = 0;
	obj = man.objs.link;
	if (obj) {
		do {
			if (iteratorIndex >= iteratorSize) {
				break;
			}
			objIterator[iteratorIndex++] = obj;

			obj = man.objs.next(&man.objs, obj);
		} while (obj && obj != man.objs.link);
	}
	iteratorIndex = 0;

#ifdef WIN_DEBUG

	if (cam) {
		res[index++] = cam->M.mx.x + cam->M.mx.y + cam->M.mx.z + cam->M.mx.w + cam->M.my.x + cam->M.my.y + cam->M.my.z + cam->M.my.w;
		res[index++] = cam->_M.Ms.mx.x + cam->_M.Ms.mx.y + cam->_M.Ms.mx.z + cam->_M.Ms.mx.w + cam->_M.Ms.my.x + cam->_M.Ms.my.y + cam->_M.Ms.my.z + cam->_M.Ms.my.w;
		res[index++] = cam->_M.M->mx.x + cam->_M.M->mx.y + cam->_M.M->mx.z + cam->_M.M->mx.w + cam->_M.M->my.x + cam->_M.M->my.y + cam->_M.M->my.z + cam->_M.M->my.w;
		res[index++] = cam->_M.Mm.mx.x + cam->_M.Mm.mx.y + cam->_M.Mm.mx.z + cam->_M.Mm.mx.w + cam->_M.Mm.my.x + cam->_M.Mm.my.y + cam->_M.Mm.my.z + cam->_M.Mm.my.w;
		res[index++] = cam->_M.Mrx.mx.x + cam->_M.Mrx.mx.y + cam->_M.Mrx.mx.z + cam->_M.Mrx.mx.w + cam->_M.Mrx.my.x + cam->_M.Mrx.my.y + cam->_M.Mrx.my.z + cam->_M.Mrx.my.w;
	}

	obj = man.objs.link;
	if (obj) {
		res[index++] = obj->M.mx.x + obj->M.mx.y + obj->M.mx.z + obj->M.mx.w + obj->M.my.x + obj->M.my.y + obj->M.my.z + obj->M.my.w;
		res[index++] = obj->_M.Ms.mx.x + obj->_M.Ms.mx.y + obj->_M.Ms.mx.z + obj->_M.Ms.mx.w + obj->_M.Ms.my.x + obj->_M.Ms.my.y + obj->_M.Ms.my.z + obj->_M.Ms.my.w;
		res[index++] = obj->_M.M->mx.x + obj->_M.M->mx.y + obj->_M.M->mx.z + obj->_M.M->mx.w + obj->_M.M->my.x + obj->_M.M->my.y + obj->_M.M->my.z + obj->_M.M->my.w;
		res[index++] = obj->_M.Mm.mx.x + obj->_M.Mm.mx.y + obj->_M.Mm.mx.z + obj->_M.Mm.mx.w + obj->_M.Mm.my.x + obj->_M.Mm.my.y + obj->_M.Mm.my.z + obj->_M.Mm.my.w;
		res[index++] = obj->_M.Mrx.mx.x + obj->_M.Mrx.mx.y + obj->_M.Mrx.mx.z + obj->_M.Mrx.mx.w + obj->_M.Mrx.my.x + obj->_M.Mrx.my.y + obj->_M.Mrx.my.z + obj->_M.Mrx.my.w;
	}

	res[index++] = 11111111111111;
	res[index++] = (DWORD)man.cams.link;
	res[index++] = iteratorSize;
	res[index++] = (DWORD)&man;
	res[index++] = man.initialized;
	res[index++] = man.objs.linkcount;
	res[index++] = (DWORD)man.objs.link;
	res[index++] = 11111111111111;

	res[index++] = (DWORD)objIterator[iteratorIndex++];
	res[index++] = (DWORD)objIterator[iteratorIndex++];
#endif
#endif

}

// Helper function for using CUDA
hipError_t rotateWithCuda(EFTYPE * res, int res_size, Manager3D * man, EFTYPE ax, EFTYPE ay, EFTYPE az)
{
	hipError_t cudaStatus;

	/////////////////////////////////////////
	//
	cudaStatus = hipMemset(dev_resf, 0, sizeof(EFTYPE)* res_size);
	if (cudaStatus != cudaStatus) {
		fprintf(stderr, "hipMemset failed!");
		goto Error7;
	}
	//�˺���������
	//<<<�鲢�������̲߳�������ÿ����ʹ�õĹ����ڴ��С��������>>>
	// Launch a kernel on the GPU with one thread for each element.
	rotateKernel << <1, 1 >> > (dev_resf, res_size, dev_man, objIterator, MAX_ITERATOR, ax, ay, az);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "rotateKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error7;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching rotateKernel!\n", cudaStatus);
		goto Error7;
	}

#ifdef WIN_DEBUG
	///////////////////////////////////////
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(res, dev_resf, res_size * sizeof(EFTYPE), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error7;
	}
	///////////////////////////////////////
	DEBUG("��ת:");
	for (int i = 0; i < MAX_ITERATOR; i++) {
		DEBUG("%.2f->", res[i]);
	}
	DEBUG("\n");

	/////////////////////////////////////////
	//
	cudaStatus = hipMemset(dev_resf, 0, sizeof(EFTYPE)* res_size);
	if (cudaStatus != cudaStatus) {
		fprintf(stderr, "hipMemset failed!");
		goto Error7;
	}
#endif
	//�˺���������
	//<<<�鲢�������̲߳�������ÿ����ʹ�õĹ����ڴ��С��������>>>
	// Launch a kernel on the GPU with one thread for each element.
	dim3    grid(THREAD_W_R, THREAD_H_R);
	normalizeKernel << <grid, 1 >> > (dev_resf, res_size, dev_man, objIterator, THREAD_W_R, 1, 1);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "normalizeKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error7;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching normalizeKernel!\n", cudaStatus);
		goto Error7;
	}

#ifdef WIN_DEBUG
	///////////////////////////////////////
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(res, dev_resf, res_size * sizeof(EFTYPE), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error7;
	}
	///////////////////////////////////////
	DEBUG("��תˢ��:");
	for (int i = 0; i < MAX_ITERATOR; i++) {
		DEBUG("%.2f->", res[i]);
	}
	DEBUG("\n");
#endif

	goto Error;
	///////////////////////////////////////
Error7:
	///////////////////////////////////////
Error:
	return cudaStatus;
}


__global__ void moveKernel(EFTYPE * res, INT size, Manager3D * _man, Obj3D ** objIterator, INT iteratorSize, EFTYPE ax, EFTYPE ay, EFTYPE az)
{
#ifdef RUN_DEVICE
	int index = 0;

	Manager3D& man = *_man;
	Obj3D * obj = NULL;
	Cam3D * cam = NULL;

	if (man.cams.link) {
		cam = man.cams.link;
		man.cams.link->_move(man.cams.link, ax, ay, az);
	}

	INT iteratorIndex = 0;
	obj = man.objs.link;
	if (obj) {
		do {
			if (iteratorIndex >= iteratorSize) {
				break;
			}
			objIterator[iteratorIndex++] = obj;

			obj = man.objs.next(&man.objs, obj);
		} while (obj && obj != man.objs.link);
	}
	iteratorIndex = 0;
#ifdef WIN_DEBUG

	if (cam) {
		res[index++] = cam->M.mx.x + cam->M.mx.y + cam->M.mx.z + cam->M.mx.w + cam->M.my.x + cam->M.my.y + cam->M.my.z + cam->M.my.w;
		res[index++] = cam->_M.Ms.mx.x + cam->_M.Ms.mx.y + cam->_M.Ms.mx.z + cam->_M.Ms.mx.w + cam->_M.Ms.my.x + cam->_M.Ms.my.y + cam->_M.Ms.my.z + cam->_M.Ms.my.w;
		res[index++] = cam->_M.M->mx.x + cam->_M.M->mx.y + cam->_M.M->mx.z + cam->_M.M->mx.w + cam->_M.M->my.x + cam->_M.M->my.y + cam->_M.M->my.z + cam->_M.M->my.w;
		res[index++] = cam->_M.Mm.mx.x + cam->_M.Mm.mx.y + cam->_M.Mm.mx.z + cam->_M.Mm.mx.w + cam->_M.Mm.my.x + cam->_M.Mm.my.y + cam->_M.Mm.my.z + cam->_M.Mm.my.w;
		res[index++] = cam->_M.Mrx.mx.x + cam->_M.Mrx.mx.y + cam->_M.Mrx.mx.z + cam->_M.Mrx.mx.w + cam->_M.Mrx.my.x + cam->_M.Mrx.my.y + cam->_M.Mrx.my.z + cam->_M.Mrx.my.w;
	}

	obj = man.objs.link;
	if (obj) {
		res[index++] = obj->M.mx.x + obj->M.mx.y + obj->M.mx.z + obj->M.mx.w + obj->M.my.x + obj->M.my.y + obj->M.my.z + obj->M.my.w;
		res[index++] = obj->_M.Ms.mx.x + obj->_M.Ms.mx.y + obj->_M.Ms.mx.z + obj->_M.Ms.mx.w + obj->_M.Ms.my.x + obj->_M.Ms.my.y + obj->_M.Ms.my.z + obj->_M.Ms.my.w;
		res[index++] = obj->_M.M->mx.x + obj->_M.M->mx.y + obj->_M.M->mx.z + obj->_M.M->mx.w + obj->_M.M->my.x + obj->_M.M->my.y + obj->_M.M->my.z + obj->_M.M->my.w;
		res[index++] = obj->_M.Mm.mx.x + obj->_M.Mm.mx.y + obj->_M.Mm.mx.z + obj->_M.Mm.mx.w + obj->_M.Mm.my.x + obj->_M.Mm.my.y + obj->_M.Mm.my.z + obj->_M.Mm.my.w;
		res[index++] = obj->_M.Mrx.mx.x + obj->_M.Mrx.mx.y + obj->_M.Mrx.mx.z + obj->_M.Mrx.mx.w + obj->_M.Mrx.my.x + obj->_M.Mrx.my.y + obj->_M.Mrx.my.z + obj->_M.Mrx.my.w;
	}

	res[index++] = 11111111111111;
	res[index++] = (DWORD)man.cams.link;
	res[index++] = iteratorSize;
	res[index++] = (DWORD)&man;
	res[index++] = man.initialized;
	res[index++] = man.objs.linkcount;
	res[index++] = (DWORD)man.objs.link;
	res[index++] = 11111111111111;

	res[index++] = (DWORD)objIterator[iteratorIndex++];
	res[index++] = (DWORD)objIterator[iteratorIndex++];
#endif
#endif

}

// Helper function for using CUDA
hipError_t moveWithCuda(EFTYPE * res, int res_size, Manager3D * man, EFTYPE ax, EFTYPE ay, EFTYPE az)
{
	hipError_t cudaStatus;

	/////////////////////////////////////////
	//
	cudaStatus = hipMemset(dev_resf, 0, sizeof(EFTYPE)* res_size);
	if (cudaStatus != cudaStatus) {
		fprintf(stderr, "hipMemset failed!");
		goto Error7;
	}
	//�˺���������
	//<<<�鲢�������̲߳�������ÿ����ʹ�õĹ����ڴ��С��������>>>
	// Launch a kernel on the GPU with one thread for each element.
	moveKernel << <1, 1 >> > (dev_resf, res_size, dev_man, objIterator, MAX_ITERATOR, ax, ay, az);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "moveKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error7;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching moveKernel!\n", cudaStatus);
		goto Error7;
	}
#ifdef WIN_DEBUG
	///////////////////////////////////////
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(res, dev_resf, res_size * sizeof(EFTYPE), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error7;
	}
	///////////////////////////////////////
	DEBUG("�ƶ�:");
	for (int i = 0; i < MAX_ITERATOR; i++) {
		DEBUG("%.2f->", res[i]);
	}
	DEBUG("\n");

	/////////////////////////////////////////
	//
	cudaStatus = hipMemset(dev_resf, 0, sizeof(EFTYPE)* res_size);
	if (cudaStatus != cudaStatus) {
		fprintf(stderr, "hipMemset failed!");
		goto Error7;
	}
#endif
	//�˺���������
	//<<<�鲢�������̲߳�������ÿ����ʹ�õĹ����ڴ��С��������>>>
	// Launch a kernel on the GPU with one thread for each element.
	dim3    grid(THREAD_W_R, THREAD_H_R);
	normalizeKernel << <grid, 1 >> > (dev_resf, res_size, dev_man, objIterator, THREAD_W_R, 1, 1);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "normalizeKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error7;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching normalizeKernel!\n", cudaStatus);
		goto Error7;
	}

#ifdef WIN_DEBUG
	///////////////////////////////////////
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(res, dev_resf, res_size * sizeof(EFTYPE), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error7;
	}
	///////////////////////////////////////
	DEBUG("�ƶ�ˢ��:");
	for (int i = 0; i < MAX_ITERATOR; i++) {
		DEBUG("%.2f->", res[i]);
	}
	DEBUG("\n");
#endif

	goto Error;
	///////////////////////////////////////
Error7:
	///////////////////////////////////////
Error :
	return cudaStatus;
}

__global__ void renderShaderVertexKernel(EFTYPE* res, INT size, Manager3D* _man, Obj3D** objIterator, INT width, INT iteratorW, INT iteratorH)
{
#ifdef RUN_DEVICE
	int index = 0;

	Manager3D& man = *_man;
	Obj3D* obj = NULL;
	Cam3D* cam = NULL;

	INT iteratorIndex = 0;
	int iteratorIndexX = blockIdx.x * iteratorW;
	int iteratorIndexY = blockIdx.y * iteratorH;

	cam = man.cams.link;
	res[99] = (DWORD)cam;

	Obj3D* _obj = NULL;
	for (int i = iteratorIndexX; i < iteratorIndexX + iteratorW; i++) {
		for (int j = iteratorIndexY; j < iteratorIndexY + iteratorH; j++) {
			Obj3D* obj = objIterator[j * width + i];

			if (obj) {
				if (!_obj) _obj = obj;
				obj->shaderVertex(obj);
				res[50 + j * width + i] = obj->debugger;
			}
		}
	}

#ifdef WIN_DEBUG

	if (cam) {
		res[index++] = cam->M.mx.x + cam->M.mx.y + cam->M.mx.z + cam->M.mx.w + cam->M.my.x + cam->M.my.y + cam->M.my.z + cam->M.my.w;
		res[index++] = cam->_M.Ms.mx.x + cam->_M.Ms.mx.y + cam->_M.Ms.mx.z + cam->_M.Ms.mx.w + cam->_M.Ms.my.x + cam->_M.Ms.my.y + cam->_M.Ms.my.z + cam->_M.Ms.my.w;
		res[index++] = cam->_M.M->mx.x + cam->_M.M->mx.y + cam->_M.M->mx.z + cam->_M.M->mx.w + cam->_M.M->my.x + cam->_M.M->my.y + cam->_M.M->my.z + cam->_M.M->my.w;
		res[index++] = cam->_M.Mm.mx.x + cam->_M.Mm.mx.y + cam->_M.Mm.mx.z + cam->_M.Mm.mx.w + cam->_M.Mm.my.x + cam->_M.Mm.my.y + cam->_M.Mm.my.z + cam->_M.Mm.my.w;
		res[index++] = cam->_M.Mrx.mx.x + cam->_M.Mrx.mx.y + cam->_M.Mrx.mx.z + cam->_M.Mrx.mx.w + cam->_M.Mrx.my.x + cam->_M.Mrx.my.y + cam->_M.Mrx.my.z + cam->_M.Mrx.my.w;
	}

	obj = man.objs.link;
	if (obj) {
		res[index++] = obj->M.mx.x + obj->M.mx.y + obj->M.mx.z + obj->M.mx.w + obj->M.my.x + obj->M.my.y + obj->M.my.z + obj->M.my.w;
		res[index++] = obj->_M.Ms.mx.x + obj->_M.Ms.mx.y + obj->_M.Ms.mx.z + obj->_M.Ms.mx.w + obj->_M.Ms.my.x + obj->_M.Ms.my.y + obj->_M.Ms.my.z + obj->_M.Ms.my.w;
		res[index++] = obj->_M.M->mx.x + obj->_M.M->mx.y + obj->_M.M->mx.z + obj->_M.M->mx.w + obj->_M.M->my.x + obj->_M.M->my.y + obj->_M.M->my.z + obj->_M.M->my.w;
		res[index++] = obj->_M.Mm.mx.x + obj->_M.Mm.mx.y + obj->_M.Mm.mx.z + obj->_M.Mm.mx.w + obj->_M.Mm.my.x + obj->_M.Mm.my.y + obj->_M.Mm.my.z + obj->_M.Mm.my.w;
		res[index++] = obj->_M.Mrx.mx.x + obj->_M.Mrx.mx.y + obj->_M.Mrx.mx.z + obj->_M.Mrx.mx.w + obj->_M.Mrx.my.x + obj->_M.Mrx.my.y + obj->_M.Mrx.my.z + obj->_M.Mrx.my.w;
	}

	res[index++] = 1111111111111;
	res[index++] = blockDim.x;
	res[index++] = iteratorIndexX;
	res[index++] = iteratorIndexY;
	res[index++] = (DWORD)&man;
	res[index++] = man.initialized;
	res[index++] = man.objs.linkcount;
	res[index++] = (DWORD)man.objs.link;
	res[index++] = 1111111111111;

	obj = man.objs.link;
	if (obj) {
		do {

			VObj* v = obj->verts.link;
			if (v) {
				do {

					res[index++] = v->v_c.x;
					res[index++] = v->v_c.y;
					res[index++] = v->v_c.z;
					res[index++] = v->v_c.w;

					v = obj->verts.next(&obj->verts, v);
				} while (v && v != obj->verts.link);
			}

			obj = man.objs.next(&man.objs, obj);
		} while (obj && obj != man.objs.link);
	}
#endif
#endif
}

__global__ void shaderVertexKernel(EFTYPE* res, INT size, Manager3D* _man, Obj3D** objIterator, INT iteratorSize, EFTYPE ax, EFTYPE ay, EFTYPE az)
{
#ifdef RUN_DEVICE
	int index = 0;

	Manager3D& man = *_man;
	Obj3D* obj = NULL;
	Cam3D* cam = NULL;

	INT iteratorIndex = 0;
	obj = man.objs.link;
	if (obj) {
		do {
			if (iteratorIndex >= iteratorSize) {
				break;
			}
			objIterator[iteratorIndex++] = obj;

			obj = man.objs.next(&man.objs, obj);
		} while (obj && obj != man.objs.link);
	}
	iteratorIndex = 0;

#ifdef WIN_DEBUG

	if (cam) {
		res[index++] = cam->M.mx.x + cam->M.mx.y + cam->M.mx.z + cam->M.mx.w + cam->M.my.x + cam->M.my.y + cam->M.my.z + cam->M.my.w;
		res[index++] = cam->_M.Ms.mx.x + cam->_M.Ms.mx.y + cam->_M.Ms.mx.z + cam->_M.Ms.mx.w + cam->_M.Ms.my.x + cam->_M.Ms.my.y + cam->_M.Ms.my.z + cam->_M.Ms.my.w;
		res[index++] = cam->_M.M->mx.x + cam->_M.M->mx.y + cam->_M.M->mx.z + cam->_M.M->mx.w + cam->_M.M->my.x + cam->_M.M->my.y + cam->_M.M->my.z + cam->_M.M->my.w;
		res[index++] = cam->_M.Mm.mx.x + cam->_M.Mm.mx.y + cam->_M.Mm.mx.z + cam->_M.Mm.mx.w + cam->_M.Mm.my.x + cam->_M.Mm.my.y + cam->_M.Mm.my.z + cam->_M.Mm.my.w;
		res[index++] = cam->_M.Mrx.mx.x + cam->_M.Mrx.mx.y + cam->_M.Mrx.mx.z + cam->_M.Mrx.mx.w + cam->_M.Mrx.my.x + cam->_M.Mrx.my.y + cam->_M.Mrx.my.z + cam->_M.Mrx.my.w;
	}

	obj = man.objs.link;
	if (obj) {
		res[index++] = obj->M.mx.x + obj->M.mx.y + obj->M.mx.z + obj->M.mx.w + obj->M.my.x + obj->M.my.y + obj->M.my.z + obj->M.my.w;
		res[index++] = obj->_M.Ms.mx.x + obj->_M.Ms.mx.y + obj->_M.Ms.mx.z + obj->_M.Ms.mx.w + obj->_M.Ms.my.x + obj->_M.Ms.my.y + obj->_M.Ms.my.z + obj->_M.Ms.my.w;
		res[index++] = obj->_M.M->mx.x + obj->_M.M->mx.y + obj->_M.M->mx.z + obj->_M.M->mx.w + obj->_M.M->my.x + obj->_M.M->my.y + obj->_M.M->my.z + obj->_M.M->my.w;
		res[index++] = obj->_M.Mm.mx.x + obj->_M.Mm.mx.y + obj->_M.Mm.mx.z + obj->_M.Mm.mx.w + obj->_M.Mm.my.x + obj->_M.Mm.my.y + obj->_M.Mm.my.z + obj->_M.Mm.my.w;
		res[index++] = obj->_M.Mrx.mx.x + obj->_M.Mrx.mx.y + obj->_M.Mrx.mx.z + obj->_M.Mrx.mx.w + obj->_M.Mrx.my.x + obj->_M.Mrx.my.y + obj->_M.Mrx.my.z + obj->_M.Mrx.my.w;
	}

	res[index++] = 11111111111111;
	res[index++] = (DWORD)man.cams.link;
	res[index++] = iteratorSize;
	res[index++] = (DWORD)&man;
	res[index++] = man.initialized;
	res[index++] = man.objs.linkcount;
	res[index++] = (DWORD)man.objs.link;
	res[index++] = 11111111111111;

	res[index++] = (DWORD)objIterator[iteratorIndex++];
	res[index++] = (DWORD)objIterator[iteratorIndex++];
#endif
#endif

}

// Helper function for using CUDA
hipError_t shaderVertexWithCuda(EFTYPE* res, int res_size, Manager3D* man, EFTYPE ax, EFTYPE ay, EFTYPE az)
{
	hipError_t cudaStatus;

	/////////////////////////////////////////
	//
	cudaStatus = hipMemset(dev_resf, 0, sizeof(EFTYPE) * res_size);
	if (cudaStatus != cudaStatus) {
		fprintf(stderr, "hipMemset failed!");
		goto Error7;
	}
	//�˺���������
	//<<<�鲢�������̲߳�������ÿ����ʹ�õĹ����ڴ��С��������>>>
	// Launch a kernel on the GPU with one thread for each element.
	shaderVertexKernel << <1, 1 >> > (dev_resf, res_size, dev_man, objIterator, MAX_ITERATOR, ax, ay, az);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "shaderVertexKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error7;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching shaderVertexKernel!\n", cudaStatus);
		goto Error7;
	}

#ifdef WIN_DEBUG
	///////////////////////////////////////
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(res, dev_resf, res_size * sizeof(EFTYPE), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error7;
	}
	///////////////////////////////////////
	DEBUG("��ת:");
	for (int i = 0; i < MAX_ITERATOR; i++) {
		DEBUG("%.2f->", res[i]);
	}
	DEBUG("\n");

	/////////////////////////////////////////
	//
	cudaStatus = hipMemset(dev_resf, 0, sizeof(EFTYPE) * res_size);
	if (cudaStatus != cudaStatus) {
		fprintf(stderr, "hipMemset failed!");
		goto Error7;
	}
#endif
	//�˺���������
	//<<<�鲢�������̲߳�������ÿ����ʹ�õĹ����ڴ��С��������>>>
	// Launch a kernel on the GPU with one thread for each element.
	dim3    grid(THREAD_W_R, THREAD_H_R);
	renderShaderVertexKernel << <grid, 1 >> > (dev_resf, res_size, dev_man, objIterator, THREAD_W_R, 1, 1);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "renderShaderVertexKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error7;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching renderShaderVertexKernel!\n", cudaStatus);
		goto Error7;
	}

#ifdef WIN_DEBUG
	///////////////////////////////////////
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(res, dev_resf, res_size * sizeof(EFTYPE), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error7;
	}
	///////////////////////////////////////
	DEBUG("������ɫ��:");
	for (int i = 0; i < MAX_ITERATOR; i++) {
		DEBUG("%.2f->", res[i]);
	}
	DEBUG("\n");
#endif

	goto Error;
	///////////////////////////////////////
Error7:
	///////////////////////////////////////
Error:
	return cudaStatus;
}

__global__ void renderReadyKernel(EFTYPE * res, INT size, Manager3D * _man, Triangles * tgIterator, INT iteratorSize, Device * device)
{
#ifdef RUN_DEVICE
	int res_index = 0;

	Manager3D& man = *_man;
	Obj3D * obj = NULL;
	Cam3D * cam = NULL;

#ifdef WIN_DEBUG
	res[res_index++] = device->width * device->height;
	res[res_index++] = 1111111111111;
	res[res_index++] = (DWORD)&man;
	res[res_index++] = man.initialized;
	res[res_index++] = man.objs.linkcount;
	res[res_index++] = (DWORD)man.objs.link;
	res[res_index++] = 1111111111111;
#endif

	VObj * _range = NULL;
	VObj * range = NULL;

	INT iteratorIndex = 0;
	obj = man.objs.link;
	if (obj) {
		int render_state = 0;//render state: 0: normal, 1: reflection, 2: refraction

		int trans_w0 = EP_MAX, trans_h0 = EP_MAX;
		int trans_w1 = -EP_MAX, trans_h1 = -EP_MAX;
		VObj * v, *v0, *v1, *vtemp;
		EPoint l1, l0, l;

		EFTYPE z;
		INT index = 0, _index = 0;
		INT xs, xe, ys, ye;
		int i, j;
		Cam3D* cam = NULL;
		Lgt3D * lgt;
		EFTYPE zz;
		EFTYPE f, t, transparent, _i, _j;
		INT line_state = 0;
		INT line_l = 0, line_r = 0;
		int inrange;
		//DWORD *_image = device->_image;
		EFTYPE* _depth = device->depth;
		DWORD* _tango = device->tango;
		DWORD* _trans = device->trans;
		EFTYPE** ___shade = device->shade;
		DWORD  ___image;
		EFTYPE depth;
		EFTYPE* __depth;
		EFTYPE* __shade;
		DWORD* __image;
		DWORD* __tango, * __trans;
		Vert3D n0, n1, n2, n3, r;
		Vert3D _n0, _n1, _n2, _n3;
		Vert3D p;


		DWORD* _image = device->threadImage[0];
		memset(_image, 0, sizeof(DWORD) * device->width * device->height);
		do {
#ifdef WIN_DEBUG
			res[res_index++] = obj->verts.linkcount;
			res[res_index++] = obj->verts_r.linkcount;
#endif

			if (iteratorIndex >= iteratorSize) {
				break;
			}

			v = obj->verts_r.link;
			// more than 3 verts
			if (v && obj->verts_r.linkcount >= 3) {
				v0 = NULL; v1 = NULL;
				int traverseCount = 0;
				do {
					traverseCount++;
					//there must be three verts
					if (v0 && v1) {
						// back face culling
						//TODO stophin �����Ȳ���
						//if (v->backface > 0)
						{

							_range = NULL;
							if (range) {
								_range = range;
							}
							else {
								_range = v;
							}
							// in range
							inrange = true;
							if (_range != v) {
								inrange = EPoint::RectIsIntersect(v->xs, v->ys, v->xe, v->ye, _range->xs, _range->ys, _range->xe, _range->ye);
								//inrange = Vert3D::CrossRect(v->xs, v->ys, v->xe, v->ye, _range->xs, _range->ys, _range->xe, _range->ye);
							}
							if (_range && inrange) {
								if (iteratorIndex >= iteratorSize) {
									break;
								}
								tgIterator[iteratorIndex].v0 = v0;
								tgIterator[iteratorIndex].v1 = v1;
								tgIterator[iteratorIndex].v = v;
								iteratorIndex++;
								v->obj = (void*)obj;


								//step1: render the triangle
								index = 0;
								xs = v->xs; xe = v->xe; ys = v->ys; ye = v->ye;
								//xs = _range == v ? v->xs : max(_range->xs, v->xs); ys = _range == v ? v->ys : max(_range->ys, v->ys);
								//xe = _range == v ? v->xe : min(_range->xe, v->xe); ye = _range == v ? v->ye : min(_range->ye, v->ye);
								//draw triangle contour
								Device::Draw_Line(_image, device->width, device->height, v0->x0, v0->y0, v1->x0, v1->y0, WHITE);
								Device::Draw_Line(_image, device->width, device->height, v1->x0, v1->y0, v->x0, v->y0, WHITE);
								Device::Draw_Line(_image, device->width, device->height, v->x0, v->y0, v0->x0, v0->y0, WHITE);


#ifdef WIN_DEBUG
								if (res_index < size) {
									res[res_index++] = ys;
									res[res_index++] = ye;
									res[res_index++] = xs;
									res[res_index++] = xe;
								}
#endif

								EFTYPE zz_f = (v->n_r.x * v->v_c.x + v->n_r.y * v->v_c.y + v->n_r.z * v->v_c.z);
								for (i = ys; i <= ye && i < device->height; i += 1) {
									cam = (Cam3D*)obj->cam;
									if (cam == NULL) {
										break;
									}

									//little trick^_^
									line_state = 0;
									line_l = 0, line_r = 0;
									if (false && device->render_linear < 0) {
										line_l = xs;
										line_r = xe;
									}
									else {
										//trick: pre-judge
										for (j = xs; j <= xe && j < device->width; j += 1) {
											__image = &_image[i * device->width + j];
											//up pulse
											if (*__image != EP_BLACK) {
												line_state++;
												if (line_state == 1) {
													line_l = j;
												}
												else {//if (line_state == 2) {
													line_r = j;
												}
												*__image = EP_BLACK;
											}
										}
									}
									EFTYPE view_h = (i - cam->offset_h) / cam->scale_h;
									for (j = line_l; j <= line_r && j < device->width; j += 1) {

										index = i * device->width + j;
										__image = &_image[index];
										if (device->render_linear < 0) {
											if (j == line_l || j == line_r) {
												*__image = obj->color;
											}
										}
										else {
											if (j >= line_l && j <= line_r) {
												*__image = obj->color;
											}
										}
										//step2: depth test
										if (*__image != EP_BLACK) {
											// get depth
											//(-n.x * ((FLOAT)j - v.x) - n.y * ((FLOAT)i - v.y)) / n.z + v->z
											n0.set((j - cam->offset_w) / cam->scale_w, view_h, 0, 1);
											//z = Vert3D::getZ(v->n_d, v->x0, v->y0, v->z, (EFTYPE)j, (EFTYPE)i);
											z = Vert3D::getZ(v->n_1_z, v->x, v->y, v->z, n0.x, n0.y);
											z *= MAX_PRECISE;
											__depth = &_depth[index];
											if (EP_ISZERO(*__depth)) {
												*__depth = z;
											}
											if (*__depth <= z) {
												*__depth = z;
											}

											if (device->render_linear < 0) {
												_image[index] = EP_BLACK;
											}
											else {
												_image[index] = EP_BLACK;
											}
										}
									}
								}
							}
						}

						if (obj->vertex_type == 1) {
							v0 = NULL;
							v1 = NULL;
						}
						else if (obj->vertex_type == 2) {
							if ((traverseCount + 1) % 2 == 0) {
								v0 = v;
								//this->v1 = this->v1;
							}
							else {
								v0 = v1;
								v1 = v;
							}
						}
						else {
							v0 = v1;
							v1 = v;
						}
					}
					else if (v0 == NULL) {
						v0 = v;
					}
					else if (v1 == NULL) {
						v1 = v;
					}

					v = obj->verts_r.next(&obj->verts_r, v);
				} while (v && v != obj->verts_r.link);
			}


			//first do objects till end
			//then do reflection and then transparent object
			if (render_state == 0) {
				obj = man.objs.next(&man.objs, obj);
				if (!(obj && obj != man.objs.link)) {
					//obj = man.tras.link;
					////do not render reflection points
					//render_state = 2;
					obj = man.refl.link;
					render_state = 1;
					if (!obj) {
						obj = man.tras.link;
						render_state = 2;
					}
				}
			}
			else if (render_state == 1) {
				obj = man.refl.next(&man.refl, obj);
				if (!(obj && obj != man.refl.link)) {
					obj = man.tras.link;
					render_state = 2;
				}

			}
			else {
				obj = man.tras.next(&man.tras, obj);
				if (!(obj && obj != man.tras.link)) {
					//render transparent after all transparent objects were done
					break;
				}
			}
		}while (obj);
	}
#ifdef WIN_DEBUG
	res[res_index++] = 99999999;
	res[res_index++] = iteratorIndex;
#endif
	res[0] = iteratorIndex;
#endif

}

#define THREAD_MUTEX_GET()
#define THREAD_MUTEX_RELEASE()

__global__ void renderKernel(EFTYPE * res, INT size, Manager3D * _man, Triangles * tgIterator, INT iteratorSize, INT grid, INT iteratorW, INT iteratorH, Device * device)
{
#ifdef RUN_DEVICE
	int res_index = 0;

	Manager3D& man = *_man;
	Obj3D * obj = NULL;
	Cam3D * cam = NULL;

	INT iteratorIndex = 0;
	int sx = blockIdx.x * iteratorW;
	int sy = blockIdx.y * iteratorH;
	int ex = blockIdx.x * iteratorW + iteratorW;
	int ey = blockIdx.y * iteratorH + iteratorH;
	int threadIdx = blockIdx.y  * blockDim.x + blockIdx.x;

	cam = man.cams.link;
	res[99] = (DWORD)cam;
	res[98] = 1000 + threadIdx;

	VObj * _range = NULL;
	VObj * range = NULL;

	INT renderIndexX = 0;
	INT renderIndexY = 0;
	int render_state = 0;
	int trans_w0 = EP_MAX, trans_h0 = EP_MAX;
	int trans_w1 = -EP_MAX, trans_h1 = -EP_MAX;
	VObj * v = NULL, *v0 = NULL, *v1 = NULL, *vtemp;
	EPoint l1, l0, l;

	EFTYPE z;
	INT index = 0, _index = 0;
	INT xs, xe, ys, ye;
	int i, j;
	Lgt3D * lgt;
	EFTYPE zz;
	EFTYPE f, t, transparent, _i, _j;
	INT line_state = 0;
	INT line_l = 0, line_r = 0;
	int inrange;

	//DWORD *_image = device->_image;
	EFTYPE *_depth = device->depth;
	DWORD *_tango = device->tango;
	DWORD *_trans = device->trans;
	EFTYPE **___shade = device->shade;
	DWORD  ___image;
	EFTYPE depth;
	EFTYPE * __depth;
	EFTYPE *__shade;
	DWORD * __image;
	DWORD * __tango, *__trans;
	Vert3D n0, n1, n2, n3, r;
	Vert3D _n0, _n1, _n2, _n3;
	Vert3D p;


	if (threadIdx >= device->threadImageCount) {
		return;
	}
	DWORD *_image = device->threadImage[threadIdx];
	memset(_image, 0, sizeof(DWORD)* device->width * device->height);

	INT width = device->width;
	INT height = device->height;
	INT render_linear = device->render_linear;
	INT render_proj = device->render_proj;
	INT render_light = device->render_light;
	INT light_count = device->light_count;


#ifdef WIN_DEBUG
	res[res_index++] = (DWORD)device->threadImage;
	res[res_index++] = (DWORD)device->threadImage[threadIdx];
	res[res_index++] = device->width * device->height;
	res[res_index++] = device->threadImageCount;
	res[res_index++] = grid;

	res[res_index++] = sx;
	res[res_index++] = sy;
	res[res_index++] = ex;
	res[res_index++] = ey;
	res[res_index++] = 99999999;
#endif

	for (renderIndexY = sy; renderIndexY < ey; renderIndexY++) {
		if (iteratorIndex >= iteratorSize) {
			break;
		}
		for (renderIndexX = sx; renderIndexX < ex; renderIndexX++) {
			iteratorIndex = renderIndexY * grid + renderIndexX;
			if (iteratorIndex >= iteratorSize) {
				break;
			}
			v0 = tgIterator[iteratorIndex].v0;
			v1 = tgIterator[iteratorIndex].v1;
			v = tgIterator[iteratorIndex].v;

			if (!v || !v0 || !v1) {
				continue;
			}
			obj = (Obj3D*)v->obj;
			if (!obj) {
				continue;
			}

			//step1: render the triangle
			index = 0;
			xs = v->xs; xe = v->xe; ys = v->ys; ye = v->ye;
			//xs = _range == v ? v->xs : max(_range->xs, v->xs); ys = _range == v ? v->ys : max(_range->ys, v->ys);
			//xe = _range == v ? v->xe : min(_range->xe, v->xe); ye = _range == v ? v->ye : min(_range->ye, v->ye);
			//draw triangle contour
			//memset(_image, 0, sizeof(DWORD) * device->width * device->height);
			//for (i = ys <= 0 ? ys : ys - 1; i <= ye + 1 && i < device->height; i += 1) {
			//	for (j = xs <= 0 ? xs : xs - 1; j <= xe + 1 && j < device->width; j += 1) {
			//		_image[i * device->width + j] = 0;
			//	}
			//}
			//Device::Draw_Line(device->tango, device->width, device->height, v0->x0, v0->y0, v1->x0, v1->y0, RED);
			//Device::Draw_Line(device->tango, device->width, device->height, v1->x0, v1->y0, v->x0, v->y0, RED);
			//Device::Draw_Line(device->tango, device->width, device->height, v->x0, v->y0, v0->x0, v0->y0, RED);
			Device::Draw_Line(_image, device->width, device->height, v0->x0, v0->y0, v1->x0, v1->y0, WHITE);
			Device::Draw_Line(_image, device->width, device->height, v1->x0, v1->y0, v->x0, v->y0, WHITE);
			Device::Draw_Line(_image, device->width, device->height, v->x0, v->y0, v0->x0, v0->y0, WHITE);

#ifdef WIN_DEBUG
			if (res_index < size) {
				res[res_index++] = ys;
				res[res_index++] = ye;
				res[res_index++] = xs;
				res[res_index++] = xe;
			}
#endif

			//get line formula
			//v0-v1
			Vert3D::GetLine(v1->v_s, v0->v_s, l1);
			//v1-v
			Vert3D::GetLine(v->v_s, v1->v_s, l);
			//v-v0
			Vert3D::GetLine(v0->v_s, v->v_s, l0);

			EFTYPE zz_f = (v->n_r.x * v->v_c.x + v->n_r.y * v->v_c.y + v->n_r.z * v->v_c.z);
			for (i = ys; i <= ye && i < device->height; i += 1) {
				cam = (Cam3D*)obj->cam;
				if (cam == NULL) {
					break;
				}

				//little trick^_^
				line_state = 0;
				line_l = xs, line_r = xe;
				if (false && device->render_linear < 0) {
					line_l = xs;
					line_r = xe;
				}
				else {
					//trick: pre-judge
					for (j = xs; j <= xe && j < device->width; j += 1) {
						__image = &_image[i * device->width + j];
						//up pulse
						if (*__image != EP_BLACK) {
							line_state++;
							if (line_state == 1) {
								line_l = j;
							}
							else { //if (line_state == 2) {
								line_r = j;
							}
							*__image = EP_BLACK;
						}
					}
				}
				//get range x
				EFTYPE __y = i;
				EFTYPE __x;
				INT _line_l1 = (INT)(l1.x * __y + l1.y);
				INT _line_l = (INT)(l.x * __y + l.y);
				INT _line_l0 = (INT)(l0.x * __y + l0.y);
				EFTYPE view_h = (i - cam->offset_h) / cam->scale_h;
				for (j = line_l; j <= line_r && j < device->width; j += 1) {

					index = i * device->width + j;
					__image = &_image[index];
					if (device->render_linear < 0) {
						if (j == line_l || j == line_r) {
							*__image = obj->color;
						}
					}
					else {
						if (j >= line_l && j <= line_r) {
							*__image = obj->color;
						}
					}
					//step2: depth test
					if (*__image != EP_BLACK) {
						// get depth
						//(-n.x * ((FLOAT)j - v.x) - n.y * ((FLOAT)i - v.y)) / n.z + v->z
						n0.set((j - cam->offset_w) / cam->scale_w, view_h, 0, 1);
						//z = Vert3D::getZ(v->n_d, v->x0, v->y0, v->z, (EFTYPE)j, (EFTYPE)i);
						z = Vert3D::getZ(v->n_1_z, v->x, v->y, v->z, n0.x, n0.y);
						zz = z * MAX_PRECISE;
						__depth = &_depth[index];
						if (EP_ISZERO(*__depth)) {
							//*__depth = z;
						}
						if ((int)*__depth == (int)zz) {
							//*__depth = z;

							__tango = &device->tango[index];
							__trans = &device->trans[index];


							//step3: render light
							//n0.set((j - cam->offset_w) / cam->scale_w, (i - cam->offset_h) / cam->scale_h, z, 1);
							n0.z = z;
							// get position
							n0 * cam->proj_1;
							zz = zz_f / (v->n_r.x * n0.x + v->n_r.y * n0.y + v->n_r.z);
							n0.x *= zz;
							n0.y *= zz;
							n0.z = zz;
							n0.w = 1;
							n1.set(n0)* cam->M_1;

							*__image = obj->getTextureColor(obj, n0, n1, n2, n3, v, NULL);

#ifdef WIN_DEBUG
							if (res_index < size) {
								res[res_index++] = (EFTYPE)*__image;
							}
#endif

							//get interpolation normal vector from 3 point of a triangle
							__x = j;
							if (1 == obj->normal_type) {
								Object3D_GetInterpolationNormalVector(v0, v1, v, __x, __y,
									line_r, line_l, _line_l1, _line_l, _line_l0,
									5, _n0, _n1, _n2, _n3);
							}
							else {
								_n0.set(v->n_r);
							}

							//calculate sumption of light factors
							lgt = man.lgts.link;
							f = 0;
							if (lgt) {
								do {
									f += lgt->getFactor(lgt, _n0, n0);

									if (device->render_light < 0) {
										break;
									}

									lgt = man.lgts.next(&man.lgts, lgt);
								} while (lgt && lgt != man.lgts.link);
							}


							//step4: render transparent
							if (!EP_ISZERO(obj->transparent)) {
								r.set(n0);
								//t = r.negative() & v->n_r;
								t = r.negative() & _n0;

								if (t < 0) t = -t;
								transparent = 1.0 / obj->transparent;
								if (transparent < 0) transparent = -transparent;

								_i = (i - obj->center_r.y) * (transparent / t) + obj->center_r.y;
								_j = (j - obj->center_r.x) * (transparent / t) + obj->center_r.x;

								if (obj->transparent < 0) {
									_i = 2 * obj->center_r.y - _i;
									_j = 2 * obj->center_r.x - _j;
								}
								if (!(_i < 0 || _i > device->height - 1 || _j < 0 || _j > device->width - 1)) {
									_index = (INT)_i * device->width + (INT)_j;
									//if (depth[_index] < z) 

									//*__trans = Light3D::multi(*__image, f);
									*__trans = Light3D_add(*__image, device->tango[_index], f);

									if (*__trans == EP_BLACK) {
										//*__trans++;
										*__trans = *__image;
									}
									//get transparent range
									if (trans_w1 < j) trans_w1 = j;
									if (trans_h1 < i) trans_h1 = i;
									if (trans_w0 > j) trans_w0 = j;
									if (trans_h0 > i) trans_h0 = i;
								}
							}
							else {
								*__image = Light3D_multi(*__image, f);
							}
							*__tango = *__image;

							//step5: render shadow map
							lgt = man.lgts.link;
							int shadeIndex = 0;
							EFTYPE * _shade = NULL;

							if (lgt) {
								do {
									if (shadeIndex >= device->light_count) {
										break;
									}
									_shade = device->shade[shadeIndex++];

									n2.set(n1) * lgt->M_1;
									cam->project(cam, n2);
									_j = (int)(n2.x * cam->scale_w + cam->offset_w), _i = (int)(n2.y * cam->scale_h + cam->offset_h);

									if (!(_i < 0 || _i > device->height - 1 || _j < 0 || _j > device->width - 1)) {
										_index = _i * device->width + _j;

										__shade = &_shade[_index];
										if (device->render_proj > 0) {
											device->tango[_index] = RED;// obj->color;
										}

										//shadow
										if (EP_GTZERO(*__shade - n2.z - 1e-1)) {
											*__tango = Light3D_multi(*__tango, f / 5);
										}
									}

									lgt = man.lgts.next(&man.lgts, lgt);
								} while (lgt && lgt != man.lgts.link);
							}

							if (device->render_proj > 0) {
								n2.set(n0)*cam->M_1 *obj->M_1* obj->M* cam->M;
								//n2.set(n1) * cam->M;
								cam->project(cam, n2);

								INT __j = (int)(n2.x * cam->scale_w + cam->offset_w), __i = (int)(n2.y * cam->scale_h + cam->offset_h);
								//Draw_Line(_tango, width, height, __j, __i, _j, _i, RED);
								_j = __j, _i = __i;

								if (!(_i < 0 || _i > device->height - 1 || _j < 0 || _j > device->width - 1)) {
									_index = _i * device->width + _j;
									device->tango[_index] = BLUE;// obj->color
								}
							}
						}

						if (device->render_linear < 0) {
							_image[index] = EP_BLACK;
						}
						else {
							_image[index] = EP_BLACK;
						}
					}
				}
			}
		}
		//render transparent after all transparent objects were done
		index = 0;
		for (i = trans_h0; i <= trans_h1 && i < device->height; i++) {
			for (j = trans_w0; j <= trans_w1 && j < device->width; j++) {
				index = i * device->width + j;
				__trans = &device->trans[index];
				if (*__trans != EP_BLACK) {
					__tango = &device->tango[index];
					*__tango = *__trans;
					*__trans = EP_BLACK;
				}
			}
		}
	}

#endif
}


// Helper function for using CUDA
hipError_t renderWithCuda(EFTYPE * res, int res_size, Manager3D * man, DWORD * image, int img_size)
{
	hipError_t cudaStatus;

	/////////////////////////////////////////
	//
	cudaStatus = hipMemset(dev_resf, 0, sizeof(EFTYPE)* res_size);
	if (cudaStatus != cudaStatus) {
		fprintf(stderr, "hipMemset failed!");
		goto Error7;
	}

	//raytracing
	if (0) {
		cudaStatus = hipMemset(tango, 0, img_size * sizeof(DWORD));
		if (cudaStatus != cudaStatus) {
			fprintf(stderr, "hipMemset failed!");
			goto Error7;
		}

		//�˺���������
		//<<<�鲢�������̲߳�������ÿ����ʹ�õĹ����ڴ��С��������>>>
		// Launch a kernel on the GPU with one thread for each element.
		dim3    grid(THREAD_W, THREAD_H);
		renderRayTracing << <grid, 1>> > (dev_resf, res_size, dev_man, dev_vman, tgIterator, THREAD_W, WIN_WIDTH / THREAD_W, WIN_HEIGHT / THREAD_H, _device, verts_pool);
		//���߳����ڻ����ڴ�̫�࣬�����ڲ���
		//dim3    grid(THREAD_W, THREAD_H);
		//renderKernel << <grid, 1 >> > (dev_resf, res_size, dev_man, tgIterator, THREAD_W, 1, 1, _device);
	}
	else {
		cudaStatus = hipMemset(tango, 0, img_size * sizeof(DWORD));
		if (cudaStatus != cudaStatus) {
			fprintf(stderr, "hipMemset failed!");
			goto Error7;
		}
		cudaStatus = hipMemset(depth, 0, img_size * sizeof(EFTYPE));
		if (cudaStatus != cudaStatus) {
			fprintf(stderr, "hipMemset failed!");
			goto Error7;
		}

		//�˺���������
		//<<<�鲢�������̲߳�������ÿ����ʹ�õĹ����ڴ��С��������>>>
		// Launch a kernel on the GPU with one thread for each element.
		renderReadyKernel << <1, 1 >> > (dev_resf, res_size, dev_man, tgIterator, MAX_ITERATOR, _device);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "renderReadyKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error7;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching renderReadyKernel!\n", cudaStatus);
			goto Error7;
		}
	#ifdef WIN_DEBUG
		///////////////////////////////////////
		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(res, dev_resf, res_size * sizeof(EFTYPE), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error7;
		}
		///////////////////////////////////////
		DEBUG("��Ⱦ:");
		for (int i = 0; i < MAX_ITERATOR; i++) {
			DEBUG("%.2f->", res[i]);
		}
		DEBUG("\n");


		/////////////////////////////////////////
		//
		cudaStatus = hipMemset(dev_resf, 0, sizeof(EFTYPE)* res_size);
		if (cudaStatus != cudaStatus) {
			fprintf(stderr, "hipMemset failed!");
			goto Error7;
		}
	#endif
		///////////////////////////////////////
		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(res, dev_resf, 1 * sizeof(EFTYPE), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error7;
		}
		INT iteratorSize = res[0];

		//�˺���������
		//<<<�鲢�������̲߳�������ÿ����ʹ�õĹ����ڴ��С��������>>>
		// Launch a kernel on the GPU with one thread for each element.
		dim3    grid(THREAD_W_R, THREAD_H_R);
		renderKernel << <grid, 1 >> > (dev_resf, res_size, dev_man, tgIterator, iteratorSize, THREAD_W_R, 1, 1, _device);
		//���߳����ڻ����ڴ�̫�࣬�����ڲ���
		//dim3    grid(THREAD_W, THREAD_H);
		//renderKernel << <grid, 1 >> > (dev_resf, res_size, dev_man, tgIterator, THREAD_W, 1, 1, _device);
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "renderKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error7;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching renderKernel!%s\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error7;
	}
//#ifdef WIN_DEBUG
	///////////////////////////////////////
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(res, dev_resf, res_size * sizeof(EFTYPE), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error7;
	}
	///////////////////////////////////////
	DEBUG("��Ⱦ���:");
	for (int i = 0; i < MAX_ITERATOR; i++) {
		DEBUG("%.2f->", res[i]);
	}
	DEBUG("\n");
//#endif

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(image, tango, img_size * sizeof(DWORD), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy image failed!");
		goto Error7;
	}
	///////////////////////////////////////
	//for (int i = 0; i < img_size; i++) {
	//	DEBUG("%d->", image[i]);
	//}
	//DEBUG("\n");

	goto Error;
	///////////////////////////////////////
Error7:
	///////////////////////////////////////
Error :
	return cudaStatus;
}


DWORD res[WIN_WIDTH * WIN_HEIGHT] = { 0 };
EFTYPE resf[MAX_ITERATOR] = { 0 };

VOID onClose() {
}

INT isInputBlocked() {
	return 0;
}

INT isresize = -1;
INT isrefresh = -1;
INT width;
INT height;
INT enter_once = 1;
VOID onPaint(HWND hWnd)
{
	if (isresize)
	{
		if (isresize != -1)
		{
			onClose();
		}
		isresize = 0;

		//
		//width = getwidth();
		//height = getheight();

		//device.Resize(width, height);

		//org.Set(width, height);
		//org /= 2;
		//scalex = org.x / 4, scaley = org.x / 4;

		//man.setCameraRange(org.x, org.y, scalex, scaley);
	}
	if (isrefresh < 1) {
		return;
	}
	isrefresh = -1;
	// Place draw code here
	EP_SetColor(EP_BLACK);
	EP_ClearDevice();
	//Render in device buffer
	//if (device.render_raytracing > 0) {
	//	if (enter_once < 0) {
	//		isrefresh = 1;
	//		//Blt buffer to window buffer
	//		DWORD * _tango = EP_GetImageBuffer();
	//		int i, j, index;
	//		for (i = 0; i < device.width; i++) {
	//			for (j = 0; j < device.height; j++){
	//				index = j *  device.width + i;
	//				if (device.raytracing[index] != EP_BLACK)
	//				{
	//					//::SetPixel(memHDC, i, j, device.tango[index]);
	//					_tango[index] = device.raytracing[index];
	//				}
	//			}
	//		}
	//		if (device.draw_line > 0) {
	//			device.drawThreadSplit();
	//		}
	//		return;
	//	}
	//	enter_once = -1;
	//	isrefresh = 1;
	//	device.ClearBeforeRayTracing();
	//	if (device.thread_count > 0) {
	//		device.RenderRayTracing(man);
	//	}
	//	else {
	//		device.RenderRayTracing_SingleThread(man);
	//	}
	//	//Blt buffer to window buffer
	//	DWORD * _tango = EP_GetImageBuffer();
	//	int i, j, index;
	//	for (i = 0; i < device.width; i++) {
	//		for (j = 0; j < device.height; j++){
	//			index = j *  device.width + i;
	//			if (device.raytracing[index] != EP_BLACK)
	//			{
	//				//::SetPixel(memHDC, i, j, device.tango[index]);
	//				_tango[index] = device.raytracing[index];
	//			}
	//		}
	//	}
	//}
	//else  {
		//if (device.draw_oct > 0) {
		//	device.drawAABB(man, &man.octTree);
		//}
		//enter_once = 1;
		//if (device.move_light > 0) {
		//	device.RenderShade(man);
		//}
		//device.ClearBeforeRender();
		//if (device.render_thread > 0) {
		//	device.RenderThread(man);
		//	if (man.changed > 0) {
		//		man.changed--;
		//		device.RenderThreadReady(man, NULL, NULL, NULL);
		//	}
		//	while (!device.IsThreadDone());
		//}
		//else {
		//	device.Render(man, NULL, NULL, NULL);
		//}
		//if (device.render_mirror > 0) {
		//	device.RenderMirror(man);
		//}
		////Blt buffer to window buffer
		//DWORD * _tango = EP_GetImageBuffer();
		//int i, j, index;
		//for (i = 0; i < device.width; i++) {
		//	for (j = 0; j < device.height; j++){
		//		index = j *  device.width + i;
		//		if (device.tango[index] != EP_BLACK)
		//		{
		//			//::SetPixel(memHDC, i, j, device.tango[index]);
		//			_tango[index] = device.tango[index];
		//		}
		//	}
		//}
	//}
	shaderVertexWithCuda(resf, MAX_ITERATOR, NULL, 0, 0, -1);
	renderWithCuda(resf, MAX_ITERATOR, NULL, res, WIN_WIDTH * WIN_HEIGHT);
	//Blt buffer to window buffer
	DWORD * _tango = EP_GetImageBuffer();
	int i, j, index;
	for (i = 0; i < device.width; i++) {
		for (j = 0; j < device.height; j++){
			index = j *  device.width + i;
			if (index >= WIN_WIDTH * WIN_HEIGHT) {
				break;
			}
			if (res[index] != EP_BLACK)
			{
				//::SetPixel(memHDC, i, j, device.tango[index]);
				_tango[index] = res[index];
			}
		}
	}

	//BitBlt(hdc, 0, 0, nWidth, nHeight, memHdc, 0, 0, SRCCOPY);
}

EFTYPE scale = 10.0;
INT is_control = 0;
VOID onScroll(FLOAT delta) {
	if (is_control) {
		if (delta > 0) {
			scale += 1.0;
		}
		else {
			scale -= 1.0;
		}
		if (scale <= 0) {
			scale = 1.0;
		}
	}
	else {
		if (device.move_light > 0) {
			if (delta > 0) {
				//man.moveLight(0, 0, scale);
			}
			else {
				//man.moveLight(0, 0, -scale);
			}
		}
		else if (device.move_trans > 0) {

			//if (delta > 0) {
			//	Obj3D * obj = man.objs.link;
			//	if (obj) {
			//		do {

			//			obj->move(0, 0, scale);


			//			obj = man.objs.next(obj);
			//		} while (obj && obj != man.objs.link);
			//	}
			//}
			//else {
			//	Obj3D * obj = man.objs.link;
			//	if (obj) {
			//		do {

			//			obj->move(0, 0, -scale);


			//			obj = man.objs.next(obj);
			//		} while (obj && obj != man.objs.link);
			//	}
			//}
		}
		else {
			if (delta > 0) {
				//man.moveCamera(0, 0, scale);
				moveWithCuda(resf, MAX_ITERATOR, NULL, 0, 0, scale);
			}
			else {
				//man.moveCamera(0, 0, -scale);
				moveWithCuda(resf, MAX_ITERATOR, NULL, 0, 0, -scale);
			}
		}

		isrefresh = 1;
	}
}

EPointF menu;
VOID onMenu(FLOAT x, FLOAT y, INT mode)
{
	if (mode == 1) // mouse down
	{
		menu.X = x;
		menu.Y = y;
	}
	else if (mode == 2) // mouse move
	{
		if (EP_NTZERO(menu.X) && EP_NTZERO(menu.Y))
		{
			if (device.move_light > 0) {
				//man.moveLight(-(x - menu.X) / scale, -(y - menu.Y) / scale, 0);
			}
			else {
				//man.moveCamera(-(x - menu.X) / scale, -(y - menu.Y) / scale, 0);
				moveWithCuda(resf, MAX_ITERATOR, NULL, -(x - menu.X) / scale, -(y - menu.Y) / scale, 0);
			}
			menu.X = x;
			menu.Y = y;

			isrefresh = 1;
		}
	}
	else	// mouse up
	{
		menu.X = 0;
		menu.Y = 0;
	}
}

EPointF drag;
VOID onDrag(FLOAT x, FLOAT y, INT mode)
{
	if (mode == 1) // mouse down
	{
		drag.X = x;
		drag.Y = y;
	}
	else if (mode == 2) // mouse move
	{
		if (EP_NTZERO(drag.X) && EP_NTZERO(drag.Y))
		{
			if (device.move_light > 0) {
				//man.rotateLight(-(y - drag.Y) / scale, (x - drag.X) / scale, 0);
			}
			else {
				//man.rotateCamera(-(y - drag.Y) / scale, (x - drag.X) / scale, 0);
				rotateWithCuda(resf, MAX_ITERATOR, NULL, -(y - drag.Y) / scale, (x - drag.X) / scale, 0);
			}
			drag.X = x;
			drag.Y = y;

			isrefresh = 1;
		}
	}
	else	// mouse up
	{
		drag.X = 0;
		drag.Y = 0;
	}
}

INT lean = 0;
VOID onTimer()
{
	//return;
	//int count = 0;
	//Obj3D * obj = man.objs.link;
	//if (obj) {
	//	//obj->rotate(0, 10, 0);
	//}
	//obj = man.objs.prev(man.objs.link);
	//if (obj) {
	//	obj->rotate(0, 10, 0);
	//	//obj->scale(1.05, 1.05, 1.05);
	//}

	//obj = man.tras.link;
	//if (obj) {
	//	do {

	//		obj->rotate(0, 10, 0);
	//		/*
	//		obj->move(0, -5, 0);

	//		if (obj->center_w.y + 30 < 0) {
	//		obj->move(0, 50, 0);
	//		}
	//		*/

	//		obj = man.tras.next(obj);
	//	} while (obj && obj != man.tras.link);
	//}
}
VOID onKeyUp(WPARAM wParam) {
	switch (wParam) {
	case VK_CONTROL:
		is_control = 0;
		break;
	case 'Q':
		if (lean) {
			//man.rotateCamera(0, 0, -lean);
			lean = 0;
		}
		break;
	case 'E':
		if (lean) {
			//man.rotateCamera(0, 0, -lean);
			lean = 0;
		}
		break;
	}

	isrefresh = 1;
}
VOID onKeyDown(WPARAM wParam)
{
	if (1)
	{
		switch (wParam)
		{
		case VK_LEFT:
			wParam = 'J';
			break;
		case VK_RIGHT:
			wParam = 'L';
			break;
		case VK_UP:
			wParam = 'I';
			break;
		case VK_DOWN:
			wParam = 'K';
			break;
		case 'A':
			wParam = VK_LEFT;
			break;
		case 'D':
			wParam = VK_RIGHT;
			break;
			//case 'W':
			//	wParam = 'Y';
			//	break;
			//case 'S':
			//	wParam = 'N';
		case 'W':
			wParam = VK_UP;
			break;
		case 'S':
			wParam = VK_DOWN;
			break;
			//case 'Q':
			//	if (lean == 0) {
			//		lean = -scale * 4;
			//		man.rotateCamera(0, 0, lean);
			//	}
			//	break;
			//case 'E':
			//	if (lean == 0) {
			//		lean = scale * 4;
			//		man.rotateCamera(0, 0, lean);
			//	}
			//	break;
		}
	}
	//Object3D * obj = (Object3D *)man.objs.link;
	//obj = cur_op;
	//if (obj == NULL) {
	//	return;
	//}
	switch (wParam)
	{
	case VK_CONTROL:
		is_control = 1;
		break;
		//case VK_LEFT:
		//	man.moveCamera(-1, 0, 0);
		//	break;
		//case VK_RIGHT:
		//	man.moveCamera(1, 0, 0);
		//	break;
		//case VK_UP:
		//	man.moveCamera(0, 1, 0);
		//	break;
		//case VK_DOWN:
		//	man.moveCamera(0, -1, 0);
		//	break;
	case VK_LEFT:
		//if (cur_op) {
		//	//cur_op->setUV(cur_op->u + 1, cur_op->v);
		//	//cur_op->move(1, 0, 0);
		//}
		break;
	case VK_RIGHT:
		//if (cur_op) {
		//	//cur_op->setUV(cur_op->u - 1, cur_op->v);
		//	//cur_op->move(-1, 0, 0);
		//}
		break;
	case VK_UP:
		//if (cur_op) {
		//	//cur_op->setUV(cur_op->u, cur_op->v + 1);
		//	//cur_op->move(0, 1, 0);
		//}
		break;
	case VK_DOWN:
		//if (cur_op) {
		//	//cur_op->setUV(cur_op->u, cur_op->v - 1);
		//	//cur_op->move(0, -1, 0);
		//}
		break;
	case 'Y':
		//man.moveCamera(0, 0, 1);
		moveWithCuda(resf, MAX_ITERATOR, NULL, 0, 0, 1);
		break;
	case 'N':
		//man.moveCamera(0, 0, -1);
		moveWithCuda(resf, MAX_ITERATOR, NULL, 0, 0, -1);
		break;
	case 'U':
		//man.rotateCamera(0, 0, -1);
		rotateWithCuda(resf, MAX_ITERATOR, NULL, 0, 0, -1);
		break;
	case 'O':
		//man.rotateCamera(0, 0, 1);
		rotateWithCuda(resf, MAX_ITERATOR, NULL, 0, 0, 1);
		break;
	case 'M':
		//man.rotateCamera(1, 0, 0);
		rotateWithCuda(resf, MAX_ITERATOR, NULL, 1, 0, 0);
		break;
	case '/':
		//man.rotateCamera(-1, 0, 0);
		rotateWithCuda(resf, MAX_ITERATOR, NULL, -1, 0, 0);
		break;
	case 'P':
		//man.nextCamera();
		//man.setCameraRange(org.x, org.y, scalex, scaley);
		break;
	case 'X':
		//device.draw_line = -device.draw_line;
		break;
	case 'T':
		//device.move_light = -device.move_light;
		break;
	case 'R':
		//device.move_trans = -device.move_trans;
		break;
	case 'L':
		//man.nextLight();
		break;
	case 'H':
		//man.lgts.link->mode = -man.lgts.link->mode;
		break;
	case 'J':
		//device.render_linear = -device.render_linear;
		break;
	case 'I':
		//device.render_proj = -device.render_proj;
		break;
	case 'K':
		//device.render_light = -device.render_light;
		break;
	case 'V':
		//device.render_raytracing = -device.render_raytracing;
		break;
	case 'A':
		//obj->move(1, 0, 0);
		break;
	case 'D':
		//obj->move(-1, 0, 0);
		break;
	case 'S':
		//obj->move(0, -1, 0);
		break;
	case 'W':
		//obj->move(0, 1, 0);
		break;
	case 'G':
		//obj->move(0, 0, -1);
		break;
	case 'F':
		//obj->move(0, 0, 1);
		break;
	case 'Q':
		//if (cur_op) {
		//	Obj3D * vobj = (Obj3D*)cur_op;
		//	do {

		//		vobj->rotate(1, 0, 0);

		//		vobj = vobj->next[1];
		//	} while (vobj && vobj != cur_op);
		//}
		break;
	case 'E':
		//if (cur_op) {
		//	Obj3D * vobj = (Obj3D*)cur_op;
		//	do {

		//		vobj->rotate(-1, 0, 0);

		//		vobj = vobj->next[1];
		//	} while (vobj && vobj != cur_op);
		//}
		break;
	case 'Z':
		//if (cur_op) {
		//	Obj3D * vobj = (Obj3D*)cur_op;
		//	do {

		//		vobj->rotate(0, 1, 0);

		//		vobj = vobj->next[1];
		//	} while (vobj && vobj != cur_op);
		//}
		break;
	case 'C':
		//if (cur_op) {
		//	Obj3D * vobj = (Obj3D*)cur_op;
		//	do {

		//		vobj->rotate(0, -1, 0);

		//		vobj = vobj->next[1];
		//	} while (vobj && vobj != cur_op);
		//}
		break;
	//case '0':
	//	cur_op->texture_type = 0;
	//	break;
	//case '1':
	//	cur_op->texture_type = 1;
	//	break;
	//case '2':
	//	cur_op->texture_type = 2;
	//	break;
	//case '3':
	//	cur_op->texture_type = 3;
	//	break;
	//case '4':
	//	cur_op->texture_type = 4;
	//	break;
	//case '5':
	//	device.draw_oct = -device.draw_oct;
	//	break;
	//case '6':
	//	device.render_thread = -device.render_thread;
	//	break;
	//case 'B':
	//	DEBUG_MODE = DEBUG_MODE >> 1;
	//	if (DEBUG_MODE == 0)
	//	{
	//		DEBUG_MODE = 0x0B;
	//	}
	//	break;
	}

	isrefresh = 1;
}

int MainLoop() {

	hipError_t cudaStatus;
	cudaStatus = initializeWithCuda(resf, MAX_ITERATOR, NULL);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "initializeWithCuda failed!");
		getch();
		return 1;
	}

	cudaStatus = rotateWithCuda(resf, MAX_ITERATOR, NULL, 0.5, 0.5, 0.5);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "rotateWithCuda failed!");
		getch();
		return 1;
	}
	cudaStatus = moveWithCuda(resf, MAX_ITERATOR, NULL, 100, 100, 100);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "moveWithCuda failed!");
		getch();
		return 1;
	}

	cudaStatus = renderWithCuda(resf, MAX_ITERATOR, NULL, res, WIN_WIDTH * WIN_HEIGHT);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "renderWithCuda failed!");
		getch();
		return 1;
	}

	//ȡ����С����ť��ʹ����С������ɴ��ڼ���
	SetWindowLong(EP_GetWnd(), GWL_STYLE, GetWindowLong(EP_GetWnd(), GWL_STYLE) & ~WS_MINIMIZEBOX);

	INT count = 0;
	char str[100];
	EP_RenderStart();
	do
	{
		onPaint(EP_GetWnd());

		if (!isInputBlocked()) {
			while (EP_MouseHit())
			{
				EP_MouseMsg msg = EP_GetMouseMsg();
				if (EP_IsWheel(msg)) {
					onScroll(msg.wheel);
				}
				if (EP_IsRight(msg)) {
					if (EP_IsDown(msg))
					{
						onMenu(msg.x, msg.y, 1);
					}
					else if (EP_IsUp(msg))
					{
						onMenu(msg.x, msg.y, 0);
					}
				}
				else if (EP_IsLeft(msg)) {
					if (EP_IsDown(msg))
					{
						onDrag(msg.x, msg.y, 1);
					}
					else if (EP_IsUp(msg))
					{
						onDrag(msg.x, msg.y, 0);
					}
				}
				if (EP_IsMove(msg))
				{
					onMenu(msg.x, msg.y, 2);
					onDrag(msg.x, msg.y, 2);
				}
			}
			if (EP_KBMsg()) {
				EP_MSG msg = EP_GetKBMsg();
				if (EP_KBIsUp(msg)) {
					onKeyUp(EP_GetKey(msg));
				}
				if (EP_KBIsDown(msg)) {
					onKeyDown(EP_GetKey(msg));
				}
				if (EP_Equal(msg, VK_ESCAPE)) {
					break;
				}
			}
		}
		else {
			EP_Delay(1000);
		}
		EP_FlushKey();
		EP_FlushMouse();

		if (count++ > 2) {
			count = 0;
			onTimer();
		}
		//delay_ms(100);
		EP_RenderFlush(10000);
		sprintf_s(str, "%s", "Engine3D");
		::SetWindowText(EP_GetWnd(), str);
	} while (1);
	onClose();
	closegraph();

	cudaStatus = unInitializeWithCuda(resf, MAX_ITERATOR, NULL);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "unInitializeWithCuda failed!");
		getch();
		return 1;
	}
}


int main()
{
	///////////////////////////


#ifdef WIN_DEBUG
	EP_Init(WIN_WIDTH, WIN_HEIGHT, 1);
#else
	EP_Init(WIN_WIDTH, WIN_HEIGHT, 0);
#endif
	//EP_Init(-1, -1);
	// Host test
#ifndef RUN_DEVICE
	_VObjPoolImp(&vobjPoolImp);
	_ObjPoolImp(&objPoolImp, &vobjPoolImp);
	_CamPoolImp(&camPoolImp);
	_LgtPoolImp(&lgtPoolImp);
	_VObjPoolImp(&vobjPoolImp);
	_OctPoolImp(&_octPoolImp, &objPoolImp, &vobjPoolImp);
	_Group3DPoolImp(&group3DPoolImp, &objPoolImp, &vobjPoolImp);
	man.vobjPoolImp = &vobjPoolImp;
	man.objPoolImp = &objPoolImp;
	man.camPoolImp = &camPoolImp;
	man.lgtPoolImp = &lgtPoolImp;
	man.octPoolImp = &_octPoolImp;
	man.group3DPoolImp = &group3DPoolImp;
	man.Init();
	Camera3D * cam = &man.addCamera(50, 50, 50, 1000, 90, 90);
	cam->_move(cam, 0, 0, -200);
	man.setCameraRange(500, 240, 126, 126);

	Object3D *_obj = &man.addObject();
	_obj->addVert(_obj, -10, -10, 10).addVert(_obj, 10, -10, 10).addVert(_obj, -10, 10, 10).addVertA(_obj, 10, 10, 10, -1)
		._move(_obj, 0, -30, -300);
	_obj = &man.addObject();
	_obj->addVert(_obj, 100, 200, 300).addVert(_obj, 400, 500, 900);

	Obj3D * obj = man.objs.link;
	if (obj) {
		do {

			VObj * v = obj->verts.link;
			if (v) {
				do {

					DEBUG("(%.2f, %.2f, %.2f, %.2f)", v->v.x, v->v.y, v->v.z, v->v.w);

					v = obj->verts.next(&obj->verts, v);
				} while (v && v != obj->verts.link);
			}

			obj = man.objs.next(&man.objs, obj);
		} while (obj && obj != man.objs.link);
	}
	DEBUG("\n");


	if (man.cams.link) {
		man.cams.link->_rotate(man.cams.link, 0.5, 0.5, 0.5);
	}
	obj = man.objs.link;
	if (obj) {
		do {

			obj->render_normalize(obj, 0);

			obj = man.objs.next(&man.objs, obj);
		} while (obj && obj != man.objs.link);
	}

	obj = man.objs.link;
	if (obj) {
		do {

			VObj * v = obj->verts.link;
			if (v) {
				do {

					DEBUG("(%.2f, %.2f, %.2f, %.2f)", v->v_c.x, v->v_c.y, v->v_c.z, v->v_c.w);

					v = obj->verts.next(&obj->verts, v);
				} while (v && v != obj->verts.link);
			}

			obj = man.objs.next(&man.objs, obj);
		} while (obj && obj != man.objs.link);
	}
	DEBUG("\n");

	if (man.cams.link) {
		man.cams.link->_move(man.cams.link, 1, 1, 1);
	}
	obj = man.objs.link;
	if (obj) {
		do {

			obj->render_normalize(obj, 0);

			obj = man.objs.next(&man.objs, obj);
		} while (obj && obj != man.objs.link);
	}

	obj = man.objs.link;
	if (obj) {
		do {

			VObj * v = obj->verts.link;
			if (v) {
				do {

					DEBUG("(%.2f, %.2f, %.2f, %.2f)", v->v_c.x, v->v_c.y, v->v_c.z, v->v_c.w);

					v = obj->verts.next(&obj->verts, v);
				} while (v && v != obj->verts.link);
			}

			obj = man.objs.next(&man.objs, obj);
		} while (obj && obj != man.objs.link);
	}
	DEBUG("\n");
	getch();
#endif
	///////////////////////////

	// Get device enum
	hipError_t cudaStatus;
	int num = 0;
	hipDeviceProp_t prop;
	cudaStatus = hipGetDeviceCount(&num);
	for (int i = 0; i < num; i++) {
		hipGetDeviceProperties(&prop, i);
	}

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		getch();
		return 1;
	}
		
	// Run Main Loop
	MainLoop();

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
		getch();
        return 1;
    }

	getch();
    return 0;
}

